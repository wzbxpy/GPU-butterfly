#include "hip/hip_runtime.h"
#include <iostream>
#include "../graph.h"
#include "../wtime.h"
#include "../util.h"
#include "butterfly-GPU.h"
#include <unistd.h>
// #include "edgeCentric.cuh"
// #include "hashCentric.cuh"
// #include <hip/hip_cooperative_groups.h>
// #include <cooperative_groups/memcpy_async.h>
// #include <cooperative_groups/reduce.h>
// #define dev 1
#define chunckSize 1
#define warpSize 32
#define FULL_MASK 0xffffffff
#define inf 0x7fffffff

using namespace std;
// using namespace cooperative_groups;

template <class T>
int initializeCudaPara(int deviceId, int numThreads, T func)
{
    hipSetDevice(deviceId);
    int numBlocksPerSm = 0;
    // Number of threads my_kernel will be launched with
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, deviceId);
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm, func, numThreads, 0);
    // cout << deviceProp.multiProcessorCount << "  " << numBlocksPerSm << endl;
    int numBlocks = deviceProp.multiProcessorCount * numBlocksPerSm;
    return numBlocks;
}

struct GPUgraph
{
    long long *beginPos;
    int *edgeList;
    int vertexCount;
    long long edgeCount;
    GPUgraph(int vertexNum, long long edgeNum)
    {
        vertexCount = vertexNum;
        edgeCount = edgeNum;
        HRR(hipMalloc(&beginPos, sizeof(long long) * (vertexCount + 1)));
        HRR(hipMalloc(&edgeList, sizeof(int) * edgeCount));
    }
    double loadGraph(int vertexNum, long long edgeNum, long long *CPU_beginPos, int *CPU_edgelist)
    {
        vertexCount = vertexNum;
        edgeCount = edgeNum;
        double startTime = wtime();
        HRR(hipMemcpy(beginPos, CPU_beginPos, sizeof(long long) * (vertexCount + 1), hipMemcpyHostToDevice));
        HRR(hipMemcpy(edgeList, CPU_edgelist, sizeof(int) * (edgeCount), hipMemcpyHostToDevice));
        return getDeltaTime(startTime);
    }
    double loadBeginPos(int vertexNum, long long *CPU_beginPos)
    {
        vertexCount = vertexNum;
        double startTime = wtime();
        HRR(hipMemcpy(beginPos, CPU_beginPos, sizeof(long long) * (vertexCount + 1), hipMemcpyHostToDevice));
        return getDeltaTime(startTime);
    }
};

__device__ void loadNextVertex(int &vertex, int *nextVertex, int &nextVertexshared)
{
    if ((vertex + 1) % chunckSize != 0)
    {
        vertex++;
    }
    else
    {
        if (threadIdx.x == 0)
            nextVertexshared = atomicAdd(nextVertex, chunckSize);
        __syncthreads();
        vertex = nextVertexshared;
    }
}

__global__ void initializeBeginPosition_GPUkernel(long long beginPosition[], long long endPosition[], GPUgraph G, int boundary, bool isFirst, bool isLast)
{
    int threadId = threadIdx.x + blockIdx.x * blockDim.x;
    if (isFirst) // The first begin position need to be initialized
    {
        for (int vertex = threadId; vertex < G.vertexCount; vertex += blockDim.x * gridDim.x)
            beginPosition[vertex] = G.beginPos[vertex];
    }
    if (isLast) // The last part of end position can be directly obtained
    {
        for (int vertex = threadId; vertex < G.vertexCount; vertex += blockDim.x * gridDim.x)
            endPosition[vertex] = G.beginPos[vertex + 1];
    }
    else
    {
        for (int vertex = threadId; vertex < G.vertexCount; vertex += blockDim.x * gridDim.x)
        {
            long long pos;
            for (pos = beginPosition[vertex]; pos < G.beginPos[vertex + 1]; pos++)
            {
                if (G.edgeList[pos] >= boundary)
                    break;
            }
            endPosition[vertex] = pos;
        }
    }
}

__global__ void
edgeCentric_GPUkernel(GPUgraph G_src, GPUgraph G_dst, unsigned long long *globalCount, int *hashTable, int startVertex, int endVertex, int partitionNum, int vertexOffsets, int *nextVertex, long long maxVertexCount, long long beginPosition[], long long endPosition[], int dstOffsets)
{
    __shared__ unsigned long long sharedCount;
    __shared__ int nextVertexshared;
    hashTable = hashTable + maxVertexCount * blockIdx.x;

    if (threadIdx.x == 0)
        sharedCount = 0;
    unsigned long long count = 0;
    for (int i = threadIdx.x; i < maxVertexCount; i += blockDim.x)
    {
        hashTable[i] = 0;
    }
    __syncthreads();

    for (int vertex = startVertex + blockIdx.x * chunckSize; vertex < endVertex;)
    {
        auto vertexDegree = G_src.beginPos[vertex + 1] - G_src.beginPos[vertex];
        // put the two hop neighbor of vertex into hash map
        for (auto oneHopNeighborID = G_src.beginPos[vertex] + threadIdx.x / 32; oneHopNeighborID < G_src.beginPos[vertex + 1]; oneHopNeighborID += 32)
        {
            int oneHopNeighbor = G_src.edgeList[oneHopNeighborID];
            int bound = vertex * partitionNum + vertexOffsets < oneHopNeighbor ? vertex * partitionNum + vertexOffsets : oneHopNeighbor;
            for (auto twoHopNeighborID = beginPosition[oneHopNeighbor] + threadIdx.x % 32; twoHopNeighborID < endPosition[oneHopNeighbor]; twoHopNeighborID += 32)
            {
                int twoHopNeighbor = G_dst.edgeList[twoHopNeighborID];
                if (twoHopNeighbor >= bound)
                    break;
                count += atomicAdd(&hashTable[((twoHopNeighbor - dstOffsets) / partitionNum)], 1);
            }
        }
        __syncthreads();

        // reset the hash map
        if (vertexDegree * vertexDegree > G_dst.vertexCount) // choose the lower costs method
        // if (1)
        {
            for (int i = threadIdx.x; i < maxVertexCount; i += blockDim.x)
            {
                hashTable[i] = 0;
            }
        }
        else
        {
            for (auto oneHopNeighborID = G_src.beginPos[vertex] + threadIdx.x / 32; oneHopNeighborID < G_src.beginPos[vertex + 1]; oneHopNeighborID += 32)
            {
                int oneHopNeighbor = G_src.edgeList[oneHopNeighborID];
                int bound = vertex * partitionNum + vertexOffsets < oneHopNeighbor ? vertex * partitionNum + vertexOffsets : oneHopNeighbor;
                for (auto twoHopNeighborID = beginPosition[oneHopNeighbor] + threadIdx.x % 32; twoHopNeighborID < endPosition[oneHopNeighbor]; twoHopNeighborID += 32)
                {
                    int twoHopNeighbor = G_dst.edgeList[twoHopNeighborID];
                    if (twoHopNeighbor >= bound)
                        break;
                    hashTable[((twoHopNeighbor - dstOffsets) / partitionNum)] = 0;
                }
            }
        }

        __syncthreads();
        loadNextVertex(vertex, nextVertex, nextVertexshared);
        // vertex += gridDim.x;
    }

    atomicAdd(&sharedCount, count);
    __syncthreads();
    if (threadIdx.x == 0)
        atomicAdd(globalCount, sharedCount);
}

int BC_edge_centric(graph *G, parameter para)
{
    double startTime, transferTime = 0, computeTime = 0, initializeTime = 0;

    int numThreads = 1024;
    int numBlocks = para.processorNum;
    int partitionNum = para.partitionNum;
    // numBlocks = 1;

    // long long *D_beginPos;
    // int *D_edgeList;
    // HRR(hipMalloc(&D_beginPos, sizeof(long long) * (G->vertexCount + 1)));
    // HRR(hipMalloc(&D_edgeList, sizeof(int) * (G->edgeCount)));
    // startTime = wtime();
    // HRR(hipMemcpy(D_beginPos, G->beginPos, sizeof(long long) * (G->vertexCount + 1), hipMemcpyHostToDevice));
    // HRR(hipMemcpy(D_edgeList, G->edgeList, sizeof(int) * (G->edgeCount), hipMemcpyHostToDevice));
    // exectionTime = getDeltaTime(startTime);
    // cout << "load graph elapsed time: " << exectionTime << endl;

    unsigned long long *globalCount;
    HRR(hipMallocManaged(&globalCount, sizeof(unsigned long long)));
    *globalCount = 0;
    int *nextVertex;
    HRR(hipMallocManaged(&nextVertex, sizeof(int)));
    int *hashTable;
    long long maxVertexCountInBatch = ceil(G->vertexCount / (double)para.batchNum / (double)partitionNum);
    HRR(hipMalloc(&hashTable, maxVertexCountInBatch * numBlocks * sizeof(int)));
    GPUgraph G_src(G->subBeginPosFirst[0].size() - 1, G->subEdgeListFirst[0].size());
    GPUgraph G_dst(G->subBeginPosSecond[0].size() - 1, G->subEdgeListSecond[0].size());

    long long *D_Position;
    HRR(hipMalloc(&D_Position, sizeof(long long) * G->vertexCount * 2));

    startTime = wtime();
    for (int i = 0; i < partitionNum; i++)
    {
        transferTime += G_src.loadGraph(G->subBeginPosFirst[i].size() - 1, G->subEdgeListFirst[i].size(), &(G->subBeginPosFirst[i][0]), &(G->subEdgeListFirst[i][0]));
        for (int j = 0; j < partitionNum; j++)
        {
            transferTime += G_dst.loadGraph(G->subBeginPosSecond[j].size() - 1, G->subEdgeListSecond[j].size(), &(G->subBeginPosSecond[j][0]), &(G->subEdgeListSecond[j][0]));
            for (int b = 0; b < para.batchNum; b++)
            {
                startTime = wtime();
                initializeBeginPosition_GPUkernel<<<numBlocks, numThreads>>>(&D_Position[(b % 2) * G->vertexCount], &D_Position[((b + 1) % 2) * G->vertexCount], G_dst, maxVertexCountInBatch * partitionNum * (b + 1), b == 0, b == para.batchNum - 1);
                HRR(hipDeviceSynchronize());
                initializeTime += getDeltaTime(startTime);
                *nextVertex = numBlocks * chunckSize;
                startTime = wtime();
                edgeCentric_GPUkernel<<<numBlocks, numThreads>>>(G_src, G_dst, globalCount, hashTable, 0, 100000, G->partitionNumSrc, i, nextVertex, maxVertexCountInBatch, &D_Position[(b % 2) * G->vertexCount], &D_Position[((b + 1) % 2) * G->vertexCount], maxVertexCountInBatch * partitionNum * b);
                HRR(hipDeviceSynchronize());
                computeTime += getDeltaTime(startTime);
                // cout << G->vertexCount << endl;
            }
        }
    }
    cout << *globalCount << ' ';
    cout << initializeTime + computeTime << " " << transferTime << endl;

    // cout << initializeTime << ' ' << computeTime * partitionNum * partitionNum << " " << transferTime * partitionNum * partitionNum << endl;

    return 0;
}

static int computeEndPosition(long long beginPos1[], long long beginPos2[], int previousVertex, int lastVertex, long long batchsize, int &breakPoint)
{
    int l = previousVertex, r = lastVertex;
    long long previouscount = beginPos1[previousVertex] + beginPos2[previousVertex];
    while (l < r)
    {
        int mid = (l + r + 1) / 2;
        if (beginPos1[mid] + beginPos2[mid] - previouscount > batchsize)
        {
            r = mid - 1;
        }
        else
        {
            l = mid;
        }
    }
    breakPoint = beginPos1[l] - beginPos1[previousVertex];
    return l;
}

__global__ void wedgeCentric_GPUkernel(long long *beginPosFirst, int *edgeListFirst, long long *beginPosSecond, int *edgeListSecond, unsigned long long *globalCount, int *hashTable, int *nextVertex, int partitionNum, long long maxVertexCount, int lastVertex, int previousVertex)
{
    __shared__ unsigned long long sharedCount;
    __shared__ int nextVertexshared;
    if (threadIdx.x == 0)
        sharedCount = 0;
    __syncthreads();
    unsigned long long count = 0;
    int threadId = threadIdx.x & 0x1f;
    int warpId = (blockDim.x * blockIdx.x + threadIdx.x) / warpSize;
    int warpDim = gridDim.x * blockDim.x / warpSize;
    long long beginPosFirstOffset = beginPosFirst[previousVertex];
    long long beginPosSecondOffset = beginPosSecond[previousVertex];
    for (int vertex = previousVertex + blockIdx.x; vertex < lastVertex;)
    {
        for (auto firstNeighborID = beginPosFirst[vertex]; firstNeighborID < beginPosFirst[vertex + 1]; firstNeighborID += blockDim.x)
        {
            int firstNeighbor = firstNeighborID + threadIdx.x < beginPosFirst[vertex + 1] ? edgeListFirst[firstNeighborID + threadIdx.x - beginPosFirstOffset] : -1;
            // int bound = vertex < firstNeighbor ? vertex : firstNeighbor;
            long long secondOffset = beginPosSecond[vertex] - beginPosSecondOffset;
            int secondDegree = beginPosSecond[vertex + 1] - beginPosSecond[vertex];
            for (auto index = 0; index < secondDegree; index += warpSize)
            {
                int secondNeighborCached = index + threadId < secondDegree ? edgeListSecond[(index + threadId) + secondOffset] : inf;
                int p = 0;
                for (auto thread = 0; thread < warpSize; thread++)
                {
                    int secondNeighbor = __shfl_sync(FULL_MASK, secondNeighborCached, thread);
                    if (secondNeighbor >= vertex)
                    {
                        p = 1;
                        break;
                    }
                    if (secondNeighbor >= firstNeighbor)
                        continue;
                    // count += (firstNeighbor / partitionNum) + (secondNeighbor / partitionNum) * maxVertexCount;
                    // hashTable[threadIdx.x + blockDim.x * blockIdx.x]++;
                    count += atomicAdd(&hashTable[(firstNeighbor / partitionNum) + (secondNeighbor / partitionNum) * maxVertexCount], 1);
                }
                if (p)
                    break;
            }
        }
        __syncthreads();
        loadNextVertex(vertex, nextVertex, nextVertexshared);
        // vertex += gridDim.x;

        // if (threadIdx.x == 0)
        // {
        //     vertex = (vertex + 1) % chunckSize != 0 ? vertex + 1 : atomicAdd(nextVertex, chunckSize);
        //     // printf("vertex:%d\n", vertex);
        // }
        // vertex = __shfl_sync(FULL_MASK, vertex, 0);
    }
    atomicAdd(&sharedCount, count);
    __syncthreads();
    if (threadIdx.x == 0)
        atomicAdd(globalCount, sharedCount);
}

int BC_wedge_centric(graph *G, parameter para)
{
    double startTime, transferTime = 0, computeTime = 0, clearTime = 0;
    int numThreads = 1024;
    int numBlocks = para.processorNum;
    // numThreads = 32;
    // numBlocks = 1;
    int partitionNum = para.partitionNum;
    GPUgraph G_first(G->subBeginPosSecond[0].size() - 1, 0);
    GPUgraph G_second(G->subBeginPosSecond[0].size() - 1, 0);
    long long maxVertexCount = ceil(G->vertexCount / (double)partitionNum);

    unsigned long long *globalCount;
    HRR(hipMallocManaged(&globalCount, sizeof(unsigned long long)));
    *globalCount = 0;
    int *nextVertex;
    HRR(hipMallocManaged(&nextVertex, sizeof(int)));
    int *hashTable;
    HRR(hipMalloc(&hashTable, sizeof(int) * maxVertexCount * maxVertexCount));
    int *edgeList;
    int batchSize = G->subEdgeListSecond[0].size() * 2 / para.batchNum + 100;
    HRR(hipMallocManaged(&edgeList, sizeof(int) * batchSize));

    for (int i = 0; i < partitionNum; i++)
    {
        for (int j = 0; j < partitionNum; j++)
        {
            // load begin position
            long long *CPUbegPos_first = &G->subBeginPosSecond[i][0];
            long long *CPUbegPos_second = &G->subBeginPosSecond[j][0];
            int *CPUedgeList_first = &G->subEdgeListSecond[i][0];
            int *CPUedgeList_second = &G->subEdgeListSecond[j][0];
            transferTime += G_first.loadBeginPos(G->subBeginPosSecond[i].size() - 1, CPUbegPos_first);
            transferTime += G_second.loadBeginPos(G->subBeginPosSecond[j].size() - 1, CPUbegPos_second);
            // for (int a = 0; a < G->subBeginPosSecond[i].size() - 1; a += 1000)
            //     cout << CPUbegPos_first[a + 1] - CPUbegPos_first[a] << endl;
            int previousEnd = 0;
            int thisEnd = 0;
            int breakPoint = 0;
            // clean the hashtable
            startTime = wtime();
            HRR(hipMemset(hashTable, 0, maxVertexCount * maxVertexCount * sizeof(int)));
            HRR(hipDeviceSynchronize());
            clearTime += getDeltaTime(startTime);
            for (auto ttt = 1;; ttt++)
            {
                thisEnd = computeEndPosition(CPUbegPos_first, CPUbegPos_second, previousEnd, G_first.vertexCount, batchSize, breakPoint);
                if (thisEnd == previousEnd)
                    break;
                *nextVertex = previousEnd + numBlocks * chunckSize;
                startTime = wtime();
                HRR(hipMemcpy(edgeList, &CPUedgeList_first[CPUbegPos_first[previousEnd]], sizeof(int) * (CPUbegPos_first[thisEnd] - CPUbegPos_first[previousEnd]), hipMemcpyHostToDevice));
                HRR(hipMemcpy(&edgeList[breakPoint], &CPUedgeList_second[CPUbegPos_second[previousEnd]], sizeof(int) * (CPUbegPos_second[thisEnd] - CPUbegPos_second[previousEnd]), hipMemcpyHostToDevice));
                transferTime += getDeltaTime(startTime);
                wedgeCentric_GPUkernel<<<numBlocks, numThreads>>>(G_first.beginPos, edgeList, G_second.beginPos, edgeList + breakPoint, globalCount, hashTable, nextVertex, partitionNum, maxVertexCount, thisEnd, previousEnd);
                HRR(hipDeviceSynchronize());
                computeTime += getDeltaTime(startTime);

                previousEnd = thisEnd;
            }
        }
    }
    cout << *globalCount << ' ';
    cout << clearTime + computeTime << ' ' << transferTime << endl;

    return 0;
}

int BC_GPU(graph *G, parameter para)
{
    hipSetDevice(1);
    // int a, b;
    // HRR(hipOccupancyMaxPotentialBlockSize(&a, &b, edgeCentric_GPUkernel));
    // cout << a << " " << b << endl;
    cout << "numblocks" << initializeCudaPara(1, 1024, edgeCentric_GPUkernel) << endl;

    if (para.varient == edgecentric)
        BC_edge_centric(G, para);
    else
        BC_wedge_centric(G, para);
}
