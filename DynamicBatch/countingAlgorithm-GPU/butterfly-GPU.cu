#include "hip/hip_runtime.h"
#include <iostream>
#include "../graph.h"
#include "../wtime.h"
#include "../util.h"
#include "butterfly-GPU.h"
#include <unistd.h>
// #include "edgeCentric.cuh"
// #include "hashCentric.cuh"
// #include <hip/hip_cooperative_groups.h>
// #include <cooperative_groups/memcpy_async.h>
// #include <cooperative_groups/reduce.h>
// #define dev 1
#define warpSize 32
#define FULL_MASK 0xffffffff
#define inf 0x7fffffff
#define MAXINT 2147483641
#define DEBUG
#define sharedSize 1024 * 8
#ifdef DEBUG
#define DBGprint(...) printf(__VA_ARGS__)
#else
#define DBGprint(...)
#endif
#define SHAREDTABLE
#define markerNum 32

using namespace std;
// using namespace cooperative_groups;
const int chunckSize = 1;
#define MAXINT 2147483641

// template <class T>
// int initializeCudaPara(int deviceId, int numThreads, T func)
// {
//     hipSetDevice(deviceId);
//     int numBlocksPerSm = 0;
//     // Number of threads my_kernel will be launched with
//     hipDeviceProp_t deviceProp;
//     hipGetDeviceProperties(&deviceProp, deviceId);
//     hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm, func, numThreads, 0);
//     // cout << deviceProp.multiProcessorCount << "  " << numBlocksPerSm << endl;
//     int numBlocks = deviceProp.multiProcessorCount * numBlocksPerSm;
//     return numBlocks;
// }

struct GPUgraph
{
    long long *beginPos;
    int *edgeList;
    int vertexCount;
    long long edgeCount;
    GPUgraph(int vertexNum, long long edgeNum)
    {
        vertexCount = vertexNum;
        edgeCount = edgeNum;
        HRR(hipMalloc(&beginPos, sizeof(long long) * (long long)(vertexCount + 1)));
        HRR(hipMalloc(&edgeList, sizeof(int) * edgeCount));
    }
    GPUgraph(string path)
    {
        graph *Gtmp = new graph;
        Gtmp->loadProperties(path);
        vertexCount = Gtmp->vertexCount;
        edgeCount = Gtmp->edgeCount;
        HRR(hipMalloc(&beginPos, sizeof(long long) * (long long)(vertexCount + 1)));
        HRR(hipMalloc(&edgeList, sizeof(int) * edgeCount));
        // delete Gtmp;
    }
    double loadGraph(int vertexNum, long long edgeNum, long long *CPU_beginPos, int *CPU_edgelist)
    {
        vertexCount = vertexNum;
        edgeCount = edgeNum;
        double startTime = wtime();
        HRR(hipMemcpy(beginPos, CPU_beginPos, sizeof(long long) * (long long)(vertexCount + 1), hipMemcpyHostToDevice));
        HRR(hipMemcpy(edgeList, CPU_edgelist, sizeof(int) * (edgeCount), hipMemcpyHostToDevice));
        return getDeltaTime(startTime);
    }
    double loadBeginPos(int vertexNum, long long *CPU_beginPos)
    {
        vertexCount = vertexNum;
        double startTime = wtime();
        HRR(hipMemcpy(beginPos, CPU_beginPos, sizeof(long long) * (long long)(vertexCount + 1), hipMemcpyHostToDevice));
        return getDeltaTime(startTime);
    }
    double loadGraphFromDisk(string path, graph *Gtmp)
    {
        double startTime = wtime();
        Gtmp->loadGraph(path);
        vertexCount = Gtmp->vertexCount;
        edgeCount = Gtmp->edgeCount;
        HRR(hipMemcpy(beginPos, Gtmp->beginPos, sizeof(long long) * (long long)(vertexCount + 1), hipMemcpyHostToDevice));
        HRR(hipMemcpy(edgeList, Gtmp->edgeList, sizeof(int) * (edgeCount), hipMemcpyHostToDevice));
        return getDeltaTime(startTime);
    }
};

__device__ void loadNextVertex(int &vertex, int *nextVertex, int &nextVertexshared, bool isFirstThread, int offsets)
{
    if ((vertex + 1 - offsets) % chunckSize != 0)
    {
        vertex++;
    }
    else
    {
        if (isFirstThread)
            nextVertexshared = atomicAdd(nextVertex, chunckSize);
        __syncthreads();
        vertex = nextVertexshared;
    }
}

__global__ void initializeBeginPosition_GPUkernel(long long beginPosition[], long long endPosition[], GPUgraph G, int boundary, bool isFirst, bool isLast, int startVertex)
{
    int threadId = threadIdx.x + blockIdx.x * blockDim.x;
    int warpId = threadId / 32;
    int threadInWarp = threadId % 32;
    // if (isFirst) // The first begin position need to be initialized
    // {
    //     for (int vertex = startVertex + threadId; vertex < G.vertexCount; vertex += blockDim.x * gridDim.x)
    //         beginPosition[vertex] = G.beginPos[vertex];
    // }
    if (isLast) // The last part of end position can be directly obtained
    {
        for (int vertex = startVertex + threadId; vertex < G.vertexCount; vertex += blockDim.x * gridDim.x)
            endPosition[vertex] = G.beginPos[vertex + 1];
    }
    else
    {
        // for (int vertex = startVertex + threadId; vertex < G.vertexCount; vertex += blockDim.x * gridDim.x)
        // {
        //     long long pos;
        //     for (pos = beginPosition[vertex]; pos < G.beginPos[vertex + 1]; pos++)
        //     {
        //         if (G.edgeList[pos] >= boundary)
        //             break;
        //     }
        //     endPosition[vertex] = pos;
        // }
        for (int vertex = startVertex + warpId; vertex < G.vertexCount; vertex += blockDim.x * gridDim.x / 32)
        {
            long long pos = beginPosition[vertex] + threadInWarp;
            int bound = boundary < vertex ? boundary : vertex;
            for (; pos < G.beginPos[vertex + 1]; pos += 32)
            {
                if (G.edgeList[pos] >= bound)
                    break;
            }
            pos = beginPosition[vertex] + __reduce_min_sync(__activemask(), int(pos - beginPosition[vertex]));
            // if (threadInWarp == 0)
            endPosition[vertex] = pos;
        }
    }
}

__global__ void edgeCentric_GPUkernel(GPUgraph G_src,
                                      GPUgraph G_dst,
                                      unsigned long long *globalCount,
                                      int *hashTable,
                                      int partitionNum,
                                      int vertexOffsets,
                                      int *nextVertex,
                                      long long maxVertexCount,
                                      long long beginPosition[],
                                      long long endPosition[],
                                      int dstOffsets,
                                      int subwarpSize,
                                      int degreeBoundForClearHashtable,
                                      int startVertex,
                                      int endVertex)
{
    __shared__ unsigned long long sharedCount;
    __shared__ int nextVertexshared;
    // __shared__ unsigned long long nextOneHopNeighborID;
    hashTable = hashTable + maxVertexCount * blockIdx.x;
    if (threadIdx.x == 0)
        sharedCount = 0;
    unsigned long long count = 0;

    // Initialize Hashtable
#ifdef SHAREDTABLE
    __shared__ int sharedHashTable[sharedSize + 1];
    for (int i = threadIdx.x; i < sharedSize; i += blockDim.x)
    {
        sharedHashTable[i] = 0;
    }
    for (int i = threadIdx.x + sharedSize; i < maxVertexCount; i += blockDim.x)
    {
        hashTable[i] = 0;
    }
#else
    for (int i = threadIdx.x; i < maxVertexCount; i += blockDim.x)
    {
        hashTable[i] = 0;
    }
#endif

    __syncthreads();

    // int subwarpSize = 2;
    int subwarpNum = blockDim.x / subwarpSize;
    for (int vertex = blockIdx.x * chunckSize + startVertex; vertex < endVertex;)
    {
        int vertexDegree = G_src.beginPos[vertex + 1] - G_src.beginPos[vertex];
        // if (vertexDegree * subwarpSize < 1024)
        // {
        //     subwarpSize *= 2;
        //     subwarpNum /= 2;
        // }
        // put the two hop neighbor of vertex into hash map
        // if (threadIdx.x == 0)
        //     nextOneHopNeighborID = G_src.beginPos[vertex] + subwarpNum;
        // __syncthreads();
        for (auto oneHopNeighborID = G_src.beginPos[vertex] + threadIdx.x / subwarpSize; oneHopNeighborID < G_src.beginPos[vertex + 1]; oneHopNeighborID += subwarpNum)
        // for (auto oneHopNeighborID = G_src.beginPos[vertex] + threadIdx.x / subwarpSize; oneHopNeighborID < G_src.beginPos[vertex + 1]; oneHopNeighborID += subwarpNum)
        {
            int oneHopNeighbor = G_src.edgeList[oneHopNeighborID];
            if (oneHopNeighbor < dstOffsets)
                continue;
            int bound = vertex * partitionNum + vertexOffsets < oneHopNeighbor ? vertex * partitionNum + vertexOffsets : oneHopNeighbor;
            for (auto twoHopNeighborID = beginPosition[oneHopNeighbor] + threadIdx.x % subwarpSize; twoHopNeighborID < endPosition[oneHopNeighbor]; twoHopNeighborID += subwarpSize)
            {
                int twoHopNeighbor = G_dst.edgeList[twoHopNeighborID];
                if (twoHopNeighbor >= bound)
                    break;
                int index = (twoHopNeighbor - dstOffsets) / partitionNum;
#ifdef SHAREDTABLE
                if (index < sharedSize)
                    count += atomicAdd(&sharedHashTable[index], 1);
                else
#endif
                    count += atomicAdd(&hashTable[index], 1);

                // hashTable[(twoHopNeighbor - dstOffsets) / partitionNum]++;
                // count++;
            }
        }
        __syncthreads();

        // reset the hash map
        // if (0)
        if (vertexDegree > degreeBoundForClearHashtable) // choose the lower costs method
        {
// hashTableShared[threadIdx.x] = 0;
#ifdef SHAREDTABLE
            for (int i = threadIdx.x; i < sharedSize; i += blockDim.x)
            {
                sharedHashTable[i] = 0;
            }
            for (int i = threadIdx.x + sharedSize; i < vertex; i += blockDim.x)
            {
                hashTable[i] = 0;
            }
#else
            for (int i = threadIdx.x; i < vertex; i += blockDim.x)
            {
                hashTable[i] = 0;
            }
#endif
        }
        else
        {
            for (auto oneHopNeighborID = G_src.beginPos[vertex] + threadIdx.x / subwarpSize; oneHopNeighborID < G_src.beginPos[vertex + 1]; oneHopNeighborID += subwarpNum)
            {
                int oneHopNeighbor = G_src.edgeList[oneHopNeighborID];
                if (oneHopNeighbor < dstOffsets)
                    continue;
                int bound = vertex * partitionNum + vertexOffsets < oneHopNeighbor ? vertex * partitionNum + vertexOffsets : oneHopNeighbor;
                for (auto twoHopNeighborID = beginPosition[oneHopNeighbor] + threadIdx.x % subwarpSize; twoHopNeighborID < endPosition[oneHopNeighbor]; twoHopNeighborID += subwarpSize)
                {
                    int twoHopNeighbor = G_dst.edgeList[twoHopNeighborID];
                    if (twoHopNeighbor >= bound)
                        break;
                    int index = (twoHopNeighbor - dstOffsets) / partitionNum;
#ifdef SHAREDTABLE
                    if (index < sharedSize)
                        sharedHashTable[index] = 0;
                    else
#endif
                        hashTable[index] = 0;
                }
            }
        }
        // if (count > 0)
        //     printf("%d %lld \n", vertex, count);
        // break;

        __syncthreads();
        loadNextVertex(vertex, nextVertex, nextVertexshared, threadIdx.x == 0, startVertex);
        // vertex += gridDim.x;
    }

    atomicAdd(&sharedCount, count);
    __syncthreads();
    if (threadIdx.x == 0)
        atomicAdd(globalCount, sharedCount);
}

struct marker
{
    int element;
    int *globalNow;
    int len;
    int bound;
};
__global__ void mergeBased(GPUgraph G_src,
                           GPUgraph G_dst,
                           unsigned long long *globalCount,
                           int partitionNum,
                           int vertexOffsets,
                           int *nextVertex,
                           int startVertex,
                           int endVertex)
{
    __shared__ unsigned long long sharedCount;
    __shared__ int nextVertexshared[32];
    if (threadIdx.x == 0)
        sharedCount = 0;
    unsigned long long count = 0;
    int warpId = threadIdx.x / 32;
    int threadId = threadIdx.x % 32;
    struct marker h[markerNum];
    for (int vertex = (blockIdx.x * 32 + warpId) * chunckSize + startVertex; vertex < endVertex;)
    // if (warpId != 0 || blockIdx.x != 0)
    //     return;
    { // first creat the marker
        int vertexDegree = G_src.beginPos[vertex + 1] - G_src.beginPos[vertex];
        int thisMarkerNum = (vertexDegree - 1) / 32 + 1;
        int markerIndex = 0;
        for (auto oneHopNeighborID = G_src.beginPos[vertex] + threadId; markerIndex < thisMarkerNum; oneHopNeighborID += 32, markerIndex++)
        {
            h[markerIndex].element = MAXINT;
            if (oneHopNeighborID >= G_src.beginPos[vertex + 1])
                break;
            auto oneHopNeighbor = G_src.edgeList[oneHopNeighborID];
            // get the first neighbor in each oneHopNeighbor's neighbor list
            h[markerIndex].bound = vertex * partitionNum + vertexOffsets < oneHopNeighbor ? vertex * partitionNum + vertexOffsets : oneHopNeighbor;

            auto start = G_dst.beginPos[oneHopNeighbor];
            auto end = G_dst.beginPos[oneHopNeighbor + 1];
            h[markerIndex].globalNow = G_dst.edgeList + start;
            h[markerIndex].len = end - start;
            if (h[markerIndex].len > 0)
            {
                int element = *(h[markerIndex].globalNow);
                if (element < h[markerIndex].bound)
                    h[markerIndex].element = element;
                else
                    h[markerIndex].len = -1;
                h[markerIndex].len--;
                h[markerIndex].globalNow++;
            }
        }
        // int previousElement = -1,cc = 1;
        // __syncwarp();
        // second pop the top element in marker and add new element from its corresponding neighbor list
        for (;;)
        {
            int element = __reduce_min_sync(FULL_MASK, h[0].element);
            for (int markerIndex = 1; markerIndex < thisMarkerNum; markerIndex++)
                element = min(element, __reduce_min_sync(FULL_MASK, h[markerIndex].element));
            if (element == MAXINT)
                break;
            int wedgeCount = 0;

            for (int markerIndex = 0; markerIndex < thisMarkerNum; markerIndex++)
            {
                int matched = element == h[markerIndex].element;
                if (matched)
                {
                    h[markerIndex].element = MAXINT;
                    if (h[markerIndex].len > 0)
                    {
                        int element = *(h[markerIndex].globalNow);
                        if (element < h[markerIndex].bound)
                            h[markerIndex].element = element;
                        else
                            h[markerIndex].len = -1;
                        h[markerIndex].len--;
                        h[markerIndex].globalNow++;
                    }
                }
                wedgeCount += __reduce_add_sync(FULL_MASK, matched);
            }
            if (threadId == 0)
            {
                count += wedgeCount * (wedgeCount - 1) / 2;
                // count += wedgeCount;
                // printf("wedge count %d count %lld \n", wedgeCount, count);
            }
            // if (threadId == 0 && blockIdx.x == 36)
            //     printf("%d %d %d\n", matched, element, vertex);
            // printf("%d %d\n", threadIdx.x, h.element);
        }
        // if (threadId == 0)
        // {
        //     int oneHopNeighbor = G_src.edgeList[G_src.beginPos[vertex]];
        //     int x = G_dst.edgeList[G_dst.beginPos[oneHopNeighbor]];
        //     int y = G_dst.edgeList[G_dst.beginPos[oneHopNeighbor + 1]];
        //     if (x == y)
        //         count++;
        // }
        // if (threadId == 0 && count > 0)
        //     printf("%d %lld\n", vertex, count);
        // break;

        // __syncthreads();
        // loadNextVertex(vertex, nextVertex, nextVertexshared[warpId], threadId == 0);
        vertex += 32 * gridDim.x;
        // if (threadIdx.x == 0 && blockIdx.x == 36)
        //     printf("%d %d %d %d\n", vertex, gridDim.x, chunckSize, dstOffsets);
        // vertex += gridDim.x;
    }
    if (threadId == 0)
        atomicAdd(&sharedCount, count);
    __syncthreads();
    if (threadIdx.x == 0)
        atomicAdd(globalCount, sharedCount);
}

__global__ void hashBased1HopPerThread(GPUgraph G_src, GPUgraph G_dst, unsigned long long *globalCount, int *hashTable, int partitionNum, int vertexOffsets, int *nextVertex, long long maxVertexCount, long long beginPosition[], long long endPosition[], int dstOffsets)
{
    __shared__ unsigned long long sharedCount;
    __shared__ int nextVertexshared;
    hashTable = hashTable + maxVertexCount * blockIdx.x;

    if (threadIdx.x == 0)
        sharedCount = 0;
    unsigned long long count = 0;

    for (int i = threadIdx.x; i < maxVertexCount; i += blockDim.x)
    {
        hashTable[i] = 0;
    }
    __syncthreads();

    for (int vertex = blockIdx.x * chunckSize + dstOffsets / partitionNum; vertex < G_src.vertexCount;)
    {
        int vertexDegree = G_src.beginPos[vertex + 1] - G_src.beginPos[vertex];
        if (vertexDegree < 1024)
            break;
        // put the two hop neighbor of vertex into hash map
        for (auto oneHopNeighborID = G_src.beginPos[vertex] + threadIdx.x; oneHopNeighborID < G_src.beginPos[vertex + 1]; oneHopNeighborID += blockDim.x)
        {
            int oneHopNeighbor = G_src.edgeList[oneHopNeighborID];
            if (oneHopNeighbor < dstOffsets)
                continue;
            int bound = vertex * partitionNum + vertexOffsets < oneHopNeighbor ? vertex * partitionNum + vertexOffsets : oneHopNeighbor;
            for (auto twoHopNeighborID = beginPosition[oneHopNeighbor]; twoHopNeighborID < endPosition[oneHopNeighbor]; twoHopNeighborID++)
            {
                int twoHopNeighbor = G_dst.edgeList[twoHopNeighborID];
                if (twoHopNeighbor >= bound)
                    break;
                count += atomicAdd(&hashTable[(twoHopNeighbor - dstOffsets) / partitionNum], 1);

                // hashTable[(twoHopNeighbor - dstOffsets) / partitionNum]++;
                // count++;
            }
        }
        __syncthreads();

        // reset the hash map
        // if (0)
        if (G_dst.edgeCount / G_dst.vertexCount > G_dst.vertexCount / vertexDegree) // choose the lower costs method
        {
            // hashTableShared[threadIdx.x] = 0;
            for (int i = threadIdx.x; i < vertex; i += blockDim.x)
            {
                hashTable[i] = 0;
            }
        }
        else
        {
            for (auto oneHopNeighborID = G_src.beginPos[vertex] + threadIdx.x; oneHopNeighborID < G_src.beginPos[vertex + 1]; oneHopNeighborID += blockDim.x)
            {
                int oneHopNeighbor = G_src.edgeList[oneHopNeighborID];
                if (oneHopNeighbor < dstOffsets)
                    continue;
                int bound = vertex * partitionNum + vertexOffsets < oneHopNeighbor ? vertex * partitionNum + vertexOffsets : oneHopNeighbor;
                for (auto twoHopNeighborID = beginPosition[oneHopNeighbor]; twoHopNeighborID < endPosition[oneHopNeighbor]; twoHopNeighborID++)
                {
                    int twoHopNeighbor = G_dst.edgeList[twoHopNeighborID];
                    if (twoHopNeighbor >= bound)
                        break;
                    hashTable[(twoHopNeighbor - dstOffsets) / partitionNum] = 0;
                }
            }
        }
        // if (count > 0)
        //     printf("%d %lld \n", vertex, count);
        // break;

        __syncthreads();
        loadNextVertex(vertex, nextVertex, nextVertexshared, threadIdx.x == 0, dstOffsets / partitionNum);
        // vertex += gridDim.x;
    }

    atomicAdd(&sharedCount, count);
    __syncthreads();
    if (threadIdx.x == 0)
        atomicAdd(globalCount, sharedCount);
}

int BC_edge_centric(graph *G, parameter para)
{
    double startTime, transferTime = 0, initializeTime = 0;
    double computeTime_block_largeWorkload = 0, computeTime_warp_smallWorkload = 0;
    double computeTime_block[100] = {0}, computeTime_warp[100] = {0};

    int numThreads = 1024;
    int numBlocks = para.processorNum;
    int partitionNum = para.partitionNum;
    int degreeBoundForClearHashtable;
    // numBlocks = 1;

    unsigned long long *globalCount;
    HRR(hipMallocManaged(&globalCount, sizeof(unsigned long long)));
    *globalCount = 0;
    int *nextVertex;
    HRR(hipMallocManaged(&nextVertex, sizeof(int)));
    int *hashTable;
    long long maxVertexCountInBatch = ceil(G->vertexCount / (double)para.batchNum / (double)partitionNum);
    HRR(hipMalloc(&hashTable, maxVertexCountInBatch * numBlocks * sizeof(int)));
    // GPUgraph G_src(G->subBeginPosFirst[0].size() - 1, G->subEdgeListFirst[0].size());
    // GPUgraph G_dst(G->subBeginPosSecond[0].size() - 1, G->subEdgeListSecond[0].size());
    GPUgraph G_src(subgraphFold(para.path, partitionNum, 0, true));
    GPUgraph G_dst(subgraphFold(para.path, partitionNum, 0, false));
    graph *Gtmp = new graph;
    Gtmp->loadGraph(subgraphFold(para.path, partitionNum, 0, false));
    Gtmp->loadGraph(subgraphFold(para.path, partitionNum, 0, true));
    int vertex32 = Gtmp->findBreakVertex(32);
    int vertex1 = Gtmp->findBreakVertex(1);
    int breakVertex[10000];
    for (int i = 1; i < 1025; i++)
    {
        breakVertex[i] = Gtmp->findBreakVertex(i);
    }

    if (para.hashRecy == adaptiveRecy)
        degreeBoundForClearHashtable = (long long)G->vertexCount * G->vertexCount / G->edgeCount;
    if (para.hashRecy == scanWedgeRecy)
        degreeBoundForClearHashtable = G->vertexCount;
    if (para.hashRecy == scanHashtableRecy)
        degreeBoundForClearHashtable = 0;
    // cout << degreeBoundForClearHashtable << " " << Gtmp->findBreakVertex(degreeBoundForClearHashtable) << endl;

    long long *D_Position;
    HRR(hipMalloc(&D_Position, sizeof(long long) * G->vertexCount * 2));

    startTime = wtime();
    for (int i = 0; i < partitionNum; i++)
    {
        transferTime += G_src.loadGraphFromDisk(subgraphFold(para.path, partitionNum, i, true), Gtmp);
        // transferTime += G_src.loadGraph(G->subBeginPosFirst[i].size() - 1, G->subEdgeListFirst[i].size(), &(G->subBeginPosFirst[i][0]), &(G->subEdgeListFirst[i][0]));
        for (int j = 0; j < partitionNum; j++)
        {
            transferTime += G_dst.loadGraphFromDisk(subgraphFold(para.path, partitionNum, j, false), Gtmp);
            // transferTime += G_dst.loadGraph(G->subBeginPosSecond[j].size() - 1, G->subEdgeListSecond[j].size(), &(G->subBeginPosSecond[j][0]), &(G->subEdgeListSecond[j][0]));
            for (int b = 0; b < para.batchNum; b++)
            {
                int dstOffsets = maxVertexCountInBatch * partitionNum * b;
                startTime = wtime();
                if (b == 0)
                    HRR(hipMemcpy(D_Position, G_dst.beginPos, sizeof(long long) * (G_dst.vertexCount + 1), hipMemcpyDeviceToDevice));
                initializeBeginPosition_GPUkernel<<<numBlocks, numThreads>>>(&D_Position[(b % 2) * G->vertexCount], &D_Position[((b + 1) % 2) * G->vertexCount], G_dst, maxVertexCountInBatch * partitionNum * (b + 1), b == 0, b == para.batchNum - 1, dstOffsets);
                HRR(hipDeviceSynchronize());
                initializeTime += getDeltaTime(startTime);
                *nextVertex = numBlocks * chunckSize + maxVertexCountInBatch * b;
                startTime = wtime();
                edgeCentric_GPUkernel<<<numBlocks, numThreads>>>(G_src,
                                                                 G_dst,
                                                                 globalCount,
                                                                 hashTable,
                                                                 partitionNum,
                                                                 i,
                                                                 nextVertex,
                                                                 maxVertexCountInBatch,
                                                                 &D_Position[(b % 2) * G->vertexCount],
                                                                 &D_Position[((b + 1) % 2) * G->vertexCount],
                                                                 dstOffsets,
                                                                 para.subwarpSize,
                                                                 degreeBoundForClearHashtable,
                                                                 maxVertexCountInBatch * b,
                                                                 vertex32);
                HRR(hipDeviceSynchronize());
                computeTime_block_largeWorkload += getDeltaTime(startTime);
                if (para.smallWorkload == blockForSmallWorkload)
                {
                    for (int degreeRange = 0; degreeRange <= markerNum; degreeRange++)
                    {
                        int startVertex = breakVertex[(degreeRange + 1) * 32];
                        startVertex = max(int(maxVertexCountInBatch * b), startVertex);
                        if (degreeRange == markerNum)
                            startVertex = (maxVertexCountInBatch * b);
                        *nextVertex = numBlocks * chunckSize + startVertex;
                        int endVertex = breakVertex[degreeRange * 32];
                        if (degreeRange == 0)
                            endVertex = breakVertex[1];
                        startTime = wtime();
                        edgeCentric_GPUkernel<<<numBlocks, numThreads>>>(G_src,
                                                                         G_dst,
                                                                         globalCount,
                                                                         hashTable,
                                                                         partitionNum,
                                                                         i,
                                                                         nextVertex,
                                                                         maxVertexCountInBatch,
                                                                         &D_Position[(b % 2) * G->vertexCount],
                                                                         &D_Position[((b + 1) % 2) * G->vertexCount],
                                                                         dstOffsets,
                                                                         para.subwarpSize,
                                                                         degreeBoundForClearHashtable,
                                                                         startVertex,
                                                                         endVertex);
                        HRR(hipDeviceSynchronize());
                        computeTime_block[degreeRange] += getDeltaTime(startTime);
                    }
                }
                // cout << i << " " << j << " " << b << " " << G->vertexCount << endl;
                // for (int xxx = G->vertexCount - 5; xxx < G->vertexCount; xxx++)
                //     cout << D_Position[(b % 2) * G->vertexCount + xxx] << ' ' << D_Position[((b + 1) % 2) * G->vertexCount + xxx] << endl;
                // cout << G->vertexCount << endl;
            }

            startTime = wtime();
            mergeBased<<<numBlocks, numThreads>>>(G_src,
                                                  G_dst,
                                                  globalCount,
                                                  partitionNum,
                                                  i,
                                                  nextVertex,
                                                  vertex32,
                                                  vertex1);
            HRR(hipDeviceSynchronize());
            computeTime_warp_smallWorkload += getDeltaTime(startTime);
            if (para.smallWorkload == blockForSmallWorkload)
            {
                for (int degreeRange = 0; degreeRange < markerNum; degreeRange++)
                {
                    int startVertex = breakVertex[(degreeRange + 1) * 32];
                    *nextVertex = numBlocks * 32 * chunckSize + startVertex;
                    int endVertex = breakVertex[degreeRange * 32];
                    if (degreeRange == 0)
                        endVertex = breakVertex[1];
                    startTime = wtime();
                    mergeBased<<<numBlocks, numThreads>>>(G_src,
                                                          G_dst,
                                                          globalCount,
                                                          partitionNum,
                                                          i,
                                                          nextVertex,
                                                          startVertex,
                                                          endVertex);
                    HRR(hipDeviceSynchronize());
                    computeTime_warp[degreeRange] += getDeltaTime(startTime);
                }
            }
        }
    }
    cout << *globalCount << ' ';
    cout << initializeTime << " " << computeTime_block_largeWorkload << " " << computeTime_warp_smallWorkload << " " << transferTime << endl;

    if (para.smallWorkload == blockForSmallWorkload)
    {
        for (int degreeRange = 0; degreeRange <= markerNum; degreeRange++)
        {
            cout << computeTime_block[degreeRange] << " " << computeTime_warp[degreeRange] << endl;
        }
    }
    // cout << initializeTime << ' ' << computeTime * partitionNum * partitionNum << " " << transferTime * partitionNum * partitionNum << endl;
    delete Gtmp;
    return 0;
}

static int computeEndPosition(long long beginPos1[], long long beginPos2[], int previousVertex, int lastVertex, long long batchsize, int &breakPoint)
{
    int l = previousVertex, r = lastVertex;
    long long previouscount = beginPos1[previousVertex] + beginPos2[previousVertex];
    while (l < r)
    {
        int mid = (l + r + 1) / 2;
        if (beginPos1[mid] + beginPos2[mid] - previouscount > batchsize)
        {
            r = mid - 1;
        }
        else
        {
            l = mid;
        }
    }
    breakPoint = beginPos1[l] - beginPos1[previousVertex];
    return l;
}

__global__ void wedgeCentric_GPUkernel(long long *beginPosFirst, int *edgeListFirst, long long *beginPosSecond, int *edgeListSecond, unsigned long long *globalCount, int *hashTable, int *nextVertex, int partitionNum, long long maxVertexCount, int lastVertex, int previousVertex)
{
    __shared__ unsigned long long sharedCount;
    __shared__ int nextVertexshared;
    if (threadIdx.x == 0)
        sharedCount = 0;
    __syncthreads();
    unsigned long long count = 0;
    int threadId = threadIdx.x & 0x1f;
    int warpId = (blockDim.x * blockIdx.x + threadIdx.x) / warpSize;
    int warpDim = gridDim.x * blockDim.x / warpSize;
    long long beginPosFirstOffset = beginPosFirst[previousVertex];
    long long beginPosSecondOffset = beginPosSecond[previousVertex];
    for (int vertex = previousVertex + blockIdx.x; vertex < lastVertex;)
    {
        for (auto firstNeighborID = beginPosFirst[vertex]; firstNeighborID < beginPosFirst[vertex + 1]; firstNeighborID += blockDim.x)
        {
            int firstNeighbor = firstNeighborID + threadIdx.x < beginPosFirst[vertex + 1] ? edgeListFirst[firstNeighborID + threadIdx.x - beginPosFirstOffset] : -1;
            // int bound = vertex < firstNeighbor ? vertex : firstNeighbor;
            long long secondOffset = beginPosSecond[vertex] - beginPosSecondOffset;
            int secondDegree = beginPosSecond[vertex + 1] - beginPosSecond[vertex];
            for (auto index = 0; index < secondDegree; index += warpSize)
            {
                int secondNeighborCached = index + threadId < secondDegree ? edgeListSecond[(index + threadId) + secondOffset] : inf;
                int p = 0;
                for (auto thread = 0; thread < warpSize; thread++)
                {
                    int secondNeighbor = __shfl_sync(FULL_MASK, secondNeighborCached, thread);
                    if (secondNeighbor >= vertex)
                    {
                        p = 1;
                        break;
                    }
                    if (secondNeighbor >= firstNeighbor)
                        continue;
                    // count += (firstNeighbor / partitionNum) + (secondNeighbor / partitionNum) * maxVertexCount;
                    // hashTable[threadIdx.x + blockDim.x * blockIdx.x]++;
                    count += atomicAdd(&hashTable[(firstNeighbor / partitionNum) + (secondNeighbor / partitionNum) * maxVertexCount], 1);
                }
                if (p)
                    break;
            }
        }
        __syncthreads();
        loadNextVertex(vertex, nextVertex, nextVertexshared, threadIdx.x == 0, previousVertex);
        // vertex += gridDim.x;

        // if (threadIdx.x == 0)
        // {
        //     vertex = (vertex + 1) % chunckSize != 0 ? vertex + 1 : atomicAdd(nextVertex, chunckSize);
        //     // printf("vertex:%d\n", vertex);
        // }
        // vertex = __shfl_sync(FULL_MASK, vertex, 0);
    }
    atomicAdd(&sharedCount, count);
    __syncthreads();
    if (threadIdx.x == 0)
        atomicAdd(globalCount, sharedCount);
}

int BC_wedge_centric(graph *G, parameter para)
{
    double startTime, transferTime = 0, computeTime = 0, clearTime = 0;
    int numThreads = 1024;
    int numBlocks = para.processorNum;
    // numThreads = 32;
    // numBlocks = 1;
    int partitionNum = para.partitionNum;
    GPUgraph G_first(G->subBeginPosSecond[0].size() - 1, 0);
    GPUgraph G_second(G->subBeginPosSecond[0].size() - 1, 0);
    long long maxVertexCount = ceil(G->vertexCount / (double)partitionNum);

    unsigned long long *globalCount;
    HRR(hipMallocManaged(&globalCount, sizeof(unsigned long long)));
    *globalCount = 0;
    int *nextVertex;
    HRR(hipMallocManaged(&nextVertex, sizeof(int)));
    int *hashTable;
    HRR(hipMalloc(&hashTable, sizeof(int) * maxVertexCount * maxVertexCount));
    int *edgeList;
    int batchSize = G->subEdgeListSecond[0].size() * 2 / para.batchNum + 100;
    HRR(hipMallocManaged(&edgeList, sizeof(int) * batchSize));

    for (int i = 0; i < partitionNum; i++)
    {
        for (int j = 0; j < partitionNum; j++)
        {
            // load begin position
            long long *CPUbegPos_first = &G->subBeginPosSecond[i][0];
            long long *CPUbegPos_second = &G->subBeginPosSecond[j][0];
            int *CPUedgeList_first = &G->subEdgeListSecond[i][0];
            int *CPUedgeList_second = &G->subEdgeListSecond[j][0];
            transferTime += G_first.loadBeginPos(G->subBeginPosSecond[i].size() - 1, CPUbegPos_first);
            transferTime += G_second.loadBeginPos(G->subBeginPosSecond[j].size() - 1, CPUbegPos_second);
            // for (int a = 0; a < G->subBeginPosSecond[i].size() - 1; a += 1000)
            //     cout << CPUbegPos_first[a + 1] - CPUbegPos_first[a] << endl;
            int previousEnd = 0;
            int thisEnd = 0;
            int breakPoint = 0;
            // clean the hashtable
            startTime = wtime();
            HRR(hipMemset(hashTable, 0, maxVertexCount * maxVertexCount * sizeof(int)));
            HRR(hipDeviceSynchronize());
            clearTime += getDeltaTime(startTime);
            for (auto ttt = 1;; ttt++)
            {
                thisEnd = computeEndPosition(CPUbegPos_first, CPUbegPos_second, previousEnd, G_first.vertexCount, batchSize, breakPoint);
                if (thisEnd == previousEnd)
                    break;
                *nextVertex = previousEnd + numBlocks * chunckSize;
                startTime = wtime();
                HRR(hipMemcpy(edgeList, &CPUedgeList_first[CPUbegPos_first[previousEnd]], sizeof(int) * (CPUbegPos_first[thisEnd] - CPUbegPos_first[previousEnd]), hipMemcpyHostToDevice));
                HRR(hipMemcpy(&edgeList[breakPoint], &CPUedgeList_second[CPUbegPos_second[previousEnd]], sizeof(int) * (CPUbegPos_second[thisEnd] - CPUbegPos_second[previousEnd]), hipMemcpyHostToDevice));
                transferTime += getDeltaTime(startTime);
                wedgeCentric_GPUkernel<<<numBlocks, numThreads>>>(G_first.beginPos, edgeList, G_second.beginPos, edgeList + breakPoint, globalCount, hashTable, nextVertex, partitionNum, maxVertexCount, thisEnd, previousEnd);
                HRR(hipDeviceSynchronize());
                computeTime += getDeltaTime(startTime);

                previousEnd = thisEnd;
            }
        }
    }
    cout << *globalCount << ' ';
    cout << clearTime << " " << computeTime << ' ' << transferTime << endl;

    return 0;
}

int BC_GPU(graph *G, parameter para)
{
    hipSetDevice(0);
    // int a, b;
    // HRR(hipOccupancyMaxPotentialBlockSize(&a, &b, edgeCentric_GPUkernel));
    // cout << a << " " << b << endl;
    // cout << "numblocks" << initializeCudaPara(1, 1024, edgeCentric_GPUkernel) << endl;

    if (para.varient == edgecentric)
        BC_edge_centric(G, para);
    else
        BC_wedge_centric(G, para);
}
