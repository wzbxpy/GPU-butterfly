#include <iostream>
#include "graph.h"
#include "wtime.h"
#include "util.h"
#include "countingAlgorithm-GPU/hashPartition.cuh"
#include "countingAlgorithm-GPU/hashCentric.cuh"
#define dev 1

using namespace std;

template <class T>
int initializeCudaPara(int deviceId, int numThreads, T func)
{
    hipSetDevice(deviceId);
    int numBlocksPerSm = 0;
    // Number of threads my_kernel will be launched with
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, deviceId);
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm, func, numThreads, 0);
    cout << deviceProp.multiProcessorCount << "  " << numBlocksPerSm << endl;
    int numBlocks = deviceProp.multiProcessorCount * numBlocksPerSm;
    return numBlocks;
}

int BC_edge_centric(graph *G)
{
    double startTime, exectionTime;

    int numThreads = 1024;
    int numBlocks = initializeCudaPara(dev, numThreads, hashCentric);
    numBlocks = 128;

    long long *D_beginPos;
    int *D_edgeList;

    HRR(hipMalloc(&D_beginPos, sizeof(long long) * (G->uCount + G->vCount + 1)));
    HRR(hipMalloc(&D_edgeList, sizeof(int) * (G->edgeCount)));
    startTime = wtime();
    HRR(hipMemcpy(D_beginPos, G->beginPos, sizeof(long long) * (G->uCount + G->vCount + 1), hipMemcpyHostToDevice));
    HRR(hipMemcpy(D_edgeList, G->edgeList, sizeof(int) * (G->edgeCount), hipMemcpyHostToDevice));
    exectionTime = wtime() - startTime;
    cout << "load graph elapsed time: " << exectionTime << endl;
    int total_size = sizeof(int) * (G->edgeCount * 2);
    int *perVertexCount;
    HRR(hipMallocManaged((void **)&perVertexCount, sizeof(int) * (G->uCount + G->vCount + 1)));

    unsigned long long *globalCount;
    HRR(hipMallocManaged(&globalCount, sizeof(unsigned long long)));
    *globalCount = 0;
    int *nextVertex;
    HRR(hipMallocManaged(&nextVertex, sizeof(int)));
    int *hashTable;
    HRR(hipMalloc(&hashTable, sizeof(int) * (G->uCount + G->vCount) * numBlocks));

    long long *D_beginPos_first;
    int *D_edgeList_first;
    long long *D_beginPos_second;
    int *D_edgeList_second;

    HRR(hipMalloc(&D_beginPos_first, sizeof(long long) * G->subBeginPosFirst[0].size()));
    HRR(hipMalloc(&D_edgeList_first, sizeof(int) * (G->subEdgeListFirst[0].size())));
    HRR(hipMalloc(&D_beginPos_second, sizeof(long long) * (G->uCount + G->vCount + 1)));
    HRR(hipMalloc(&D_edgeList_second, sizeof(int) * (G->subEdgeListSecond[0].size())));

    startTime = wtime();
    double transferTime = 0, computeTime = 0;
    for (int j = 0; j < G->partitionNum; j++)
    // for (int j = 0; j < 1; j++)
    {

        HRR(hipMemcpy(D_beginPos_first, &(G->subBeginPosFirst[j][0]), sizeof(long long) * (G->subBeginPosFirst[j].size()), hipMemcpyHostToDevice));
        HRR(hipMemcpy(D_edgeList_first, &(G->subEdgeListFirst[j][0]), sizeof(int) * (G->subEdgeListFirst[j].size()), hipMemcpyHostToDevice));
        for (int i = 0; i < G->partitionNum; i++)
        {
            // cout << i << ' ' << j << endl;
            *nextVertex = numBlocks;
            startTime = wtime();
            HRR(hipMemcpy(D_beginPos_second, &(G->subBeginPosSecond[i][0]), sizeof(long long) * (G->subBeginPosSecond[i].size()), hipMemcpyHostToDevice));
            HRR(hipMemcpy(D_edgeList_second, &(G->subEdgeListSecond[i][0]), sizeof(int) * (G->subEdgeListSecond[i].size()), hipMemcpyHostToDevice));
            // *globalCount = 0;
            transferTime += wtime() - startTime;
            startTime = wtime();
            hashPartition<<<numBlocks, numThreads>>>(D_beginPos_first, D_edgeList_first, D_beginPos_second, D_edgeList_second, globalCount, perVertexCount, hashTable, 0, G->subBeginPosFirst[j].size() - 1, G->length, G->partitionNum, j, nextVertex);
            HRR(hipDeviceSynchronize());
            computeTime += wtime() - startTime;
            // cout << G->uCount + G->vCount << endl;
        }
    }
    cout << *globalCount << endl;
    exectionTime = wtime() - startTime;
    cout << transferTime << ' ' << computeTime << endl;
    // cout << *globalCount << ' ' << exectionTime << endl;

    cout << endl;

    // HRR(hipMemcpy((void **)&host_list,(void **)&Sorted_List,sizeof(int)*(G->edgeCount), hipMemcpyDeviceToHost));

    HRR(hipFree(D_beginPos));
    HRR(hipFree(D_edgeList));

    // delete(perVertexCount);
    return 0;
}

int BC_wedge_centric(graph *G)
{

    double startTime, exectionTime;

    long long *D_beginPos;
    int *D_edgeList;
    int numThreads = 1024;
    int numBlocks = initializeCudaPara(dev, numThreads, hashCentric);

    HRR(hipMalloc(&D_beginPos, sizeof(long long) * (G->uCount + G->vCount + 1)));
    HRR(hipMalloc(&D_edgeList, sizeof(int) * (G->edgeCount)));
    startTime = wtime();
    HRR(hipMemcpy(D_beginPos, G->beginPos, sizeof(long long) * (G->uCount + G->vCount + 1), hipMemcpyHostToDevice));
    HRR(hipMemcpy(D_edgeList, G->edgeList, sizeof(int) * (G->edgeCount), hipMemcpyHostToDevice));
    exectionTime = wtime() - startTime;
    cout << "load graph elapsed time: " << exectionTime << endl;
    unsigned long long *globalCount;
    HRR(hipMallocManaged(&globalCount, sizeof(unsigned long long)));
    *globalCount = 0;
    int *nextVertex;
    HRR(hipMallocManaged(&nextVertex, sizeof(int)));
    int *hashTable;
    HRR(hipMalloc(&hashTable, sizeof(int) * G->length * G->length));

    long long *D_beginPos_first;
    int *D_edgeList_first;
    long long *D_beginPos_second;
    int *D_edgeList_second;

    HRR(hipMalloc(&D_beginPos_first, sizeof(long long) * (G->uCount + G->vCount + 1)));
    HRR(hipMalloc(&D_edgeList_first, sizeof(int) * (G->subEdgeListSecond[0].size())));
    HRR(hipMalloc(&D_beginPos_second, sizeof(long long) * (G->uCount + G->vCount + 1)));
    HRR(hipMalloc(&D_edgeList_second, sizeof(int) * (G->subEdgeListSecond[0].size())));

    cout << G->vertexCount / 100 << " number of vetrex" << endl;
    *globalCount = 0;
    startTime = wtime();
    double transferTime = 0, computeTime = 0;
    for (int i = 0; i < G->partitionNum; i++)
    {
        HRR(hipMemcpy(D_beginPos_first, &(G->subBeginPosSecond[i][0]), sizeof(long long) * (G->subBeginPosSecond[i].size()), hipMemcpyHostToDevice));
        HRR(hipMemcpy(D_edgeList_first, &(G->subEdgeListSecond[i][0]), sizeof(int) * (G->subEdgeListSecond[i].size()), hipMemcpyHostToDevice));
        for (int j = 0; j < G->partitionNum; j++)
        {
            // cout << i << ' ' << j << endl;
            *nextVertex = numBlocks;
            startTime = wtime();
            HRR(hipMemcpy(D_beginPos_second, &(G->subBeginPosSecond[j][0]), sizeof(long long) * (G->subBeginPosSecond[j].size()), hipMemcpyHostToDevice));
            HRR(hipMemcpy(D_edgeList_second, &(G->subEdgeListSecond[j][0]), sizeof(int) * (G->subEdgeListSecond[j].size()), hipMemcpyHostToDevice));
            // *globalCount = 0;
            transferTime += wtime() - startTime;
            startTime = wtime();
            // clearHashTable<<<G->length, 1024>>>(hashTable, G->length);
            // HRR(hipDeviceSynchronize());
            int startVertex = 0;
            void *kernelArgs[] = {&D_beginPos_first, &D_edgeList_first, &D_beginPos_second, &D_edgeList_second, &globalCount, &hashTable, &startVertex, &G->vertexCount, &G->length, &G->partitionNum};
            hipLaunchCooperativeKernel((void *)hashCentric, numBlocks, numThreads, kernelArgs);
            // hashCentric<<<numBlocks, 1024>>>(D_beginPos_first, D_edgeList_first, D_beginPos_second, D_edgeList_second, globalCount, hashTable, 0, G->uCount + G->vCount, G->length, G->partitionNum);
            HRR(hipDeviceSynchronize());
            computeTime += wtime() - startTime;
            // cout << *globalCount << endl;
            // cout << G->uCount + G->vCount << endl;
        }
    }
    cout << *globalCount << endl;
    exectionTime = wtime() - startTime;
    cout << transferTime << ' ' << computeTime << endl;
    // cout << *globalCount << ' ' << exectionTime << endl;

    HRR(hipFree(D_beginPos));
    HRR(hipFree(D_edgeList));

    return 0;
}

int BC_GPU(graph *G, bool isEdgeCentric)
{
    if (isEdgeCentric)
        BC_edge_centric(G);
    else
        BC_wedge_centric(G);
}