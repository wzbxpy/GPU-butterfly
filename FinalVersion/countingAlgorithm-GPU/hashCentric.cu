#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/memcpy_async.h>
#include <cooperative_groups/reduce.h>
using namespace std;
using namespace cooperative_groups;
#define warpSize 32

__global__ void hashCentric(long long *beginPosFirst, int *edgeListFirst, long long *beginPosSecond, int *edgeListSecond, unsigned long long *globalCount, int *hashTable, int startVertex, int endVertex, int length, int partitionNum)
{
    for (int j = blockIdx.x; j < length; j += gridDim.x)
        for (int i = threadIdx.x + j * (length); i < (1 + j) * (length); i += blockDim.x)
            hashTable[i] = 0;
    auto grid = this_grid();
    grid.sync();
    __shared__ unsigned long long sharedCount;
    if (threadIdx.x == 0)
        sharedCount = 0;
    unsigned long long count = 0;
    int threadId = threadIdx.x % warpSize;
    int warpId = (blockDim.x * blockIdx.x + threadIdx.x) / warpSize;
    int warpDim = gridDim.x * blockDim.x / warpSize;
    for (int vertex = startVertex + warpId; vertex < endVertex; vertex += warpDim)
    {
        // if (threadIdx.x + blockIdx.x == 0)
        //     printf("%d\n", vertex);
        for (int firstNeighborID = beginPosFirst[vertex] + threadId; firstNeighborID < beginPosFirst[vertex + 1]; firstNeighborID += warpSize)
        {
            int firstNeighbor = edgeListFirst[firstNeighborID];
            int bound = vertex < firstNeighbor ? vertex : firstNeighbor;
            for (int secondNeighborID = beginPosSecond[vertex]; secondNeighborID < beginPosSecond[vertex + 1]; secondNeighborID += 1)
            {
                int secondNeighbor = edgeListSecond[secondNeighborID];
                if (secondNeighbor >= bound)
                    break;
                // if (firstNeighbor >= length || secondNeighbor >= length)
                //     printf("%d %d\n", firstNeighbor, secondNeighbor);
                count += atomicAdd(&hashTable[firstNeighbor / partitionNum * length + secondNeighbor / partitionNum], 1);
            }
        }
        __syncthreads();
    }
    atomicAdd(&sharedCount, count);
    __syncthreads();
    if (threadIdx.x == 0)
        atomicAdd(globalCount, sharedCount);
}

__global__ void clearHashTable(int *hashTable, int length)
{
    for (int j = blockIdx.x; j < length; j += gridDim.x)
        for (int i = threadIdx.x + j * (length); i < (1 + j) * (length); i += blockDim.x)
            hashTable[i] = 0;
}