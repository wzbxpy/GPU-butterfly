#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/memcpy_async.h>
#include <cooperative_groups/reduce.h>
// #include <cooperative_groups/scan.h>
using namespace std;
using namespace cooperative_groups;
// #define subgroupSize 32
#define dynamic_scheduling

__global__ void hashPartition(long long *beginPosFirst, int *edgeListFirst, long long *beginPosSecond, int *edgeListSecond, unsigned long long *globalCount, int *perVertexCount, int *hashTable, int startVertex, int endVertex, int length, int partitionNum, int vertexOffsets, int *nextVertex)
{
    __shared__ unsigned long long sharedCount;
#ifdef dynamic_scheduling
    __shared__ int nextVertexshared;
#endif
    if (threadIdx.x == 0)
        sharedCount = 0;
    unsigned long long count = 0;
    for (int i = threadIdx.x + blockIdx.x * (length); i < (1 + blockIdx.x) * (length); i += blockDim.x)
    {
        hashTable[i] = 0;
    }
    __syncthreads();

#ifdef dynamic_scheduling
    for (int vertex = startVertex + blockIdx.x; vertex < endVertex;)
    {
        if (threadIdx.x == 0)
            nextVertexshared = atomicAdd(nextVertex, 1);
        __syncthreads();
        vertex = nextVertexshared;
#else
    for (int vertex = startVertex + blockIdx.x; vertex < endVertex; vertex += gridDim.x)
    {
#endif
        int vertexDegree = beginPosFirst[vertex + 1] - beginPosFirst[vertex];

        // put the two hop neighbor of vertex into hash map

        for (int oneHopNeighborID = beginPosFirst[vertex] + threadIdx.x / 32; oneHopNeighborID < beginPosFirst[vertex + 1]; oneHopNeighborID += 32)
        {
            int oneHopNeighbor = edgeListFirst[oneHopNeighborID];
            int bound = vertex * partitionNum + vertexOffsets < oneHopNeighbor ? vertex * partitionNum + vertexOffsets : oneHopNeighbor;
            for (int twoHopNeighborID = beginPosSecond[oneHopNeighbor] + threadIdx.x % 32; twoHopNeighborID < beginPosSecond[oneHopNeighbor + 1]; twoHopNeighborID += 32)
            {
                int twoHopNeighbor = edgeListSecond[twoHopNeighborID];
                if (twoHopNeighbor >= bound)
                    break;
                count += atomicAdd(&hashTable[(twoHopNeighbor / partitionNum) + blockIdx.x * (length)], 1);
            }
        }
        // thisBlock.sync();
        // this_thread_block().sync();
        __syncthreads();
        // if (threadIdx.x + blockIdx.x == 0)
        //     printf("%d thread num\n", thisBlock.size());

        // reset the hash map
        if (vertexDegree * vertexDegree > length) //choose the lower costs method
        // if (1)
        {
            int start = 0, end = length;
            start += blockIdx.x * (length), end += blockIdx.x * (length);
            for (int i = start + threadIdx.x; i < end; i += blockDim.x)
            {
                hashTable[i] = 0;
            }
        }
        else
        {
            for (int oneHopNeighborID = beginPosFirst[vertex] + threadIdx.x / 32; oneHopNeighborID < beginPosFirst[vertex + 1]; oneHopNeighborID += 32)
            {
                int oneHopNeighbor = edgeListFirst[oneHopNeighborID];
                int bound = vertex * partitionNum + vertexOffsets < oneHopNeighbor ? vertex * partitionNum + vertexOffsets : oneHopNeighbor;
                for (int twoHopNeighborID = beginPosSecond[oneHopNeighbor] + threadIdx.x % 32; twoHopNeighborID < beginPosSecond[oneHopNeighbor + 1]; twoHopNeighborID += 32)
                {
                    int twoHopNeighbor = edgeListSecond[twoHopNeighborID];
                    if (twoHopNeighbor >= bound)
                        break;
                    hashTable[(twoHopNeighbor / partitionNum) + blockIdx.x * (length)] = 0;
                }
            }
        }
        __syncthreads();
        // this_thread_block().sync();
    }

    atomicAdd(&sharedCount, count);
    __syncthreads();
    if (threadIdx.x == 0)
        atomicAdd(globalCount, sharedCount);
}