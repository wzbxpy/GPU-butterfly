#include <iostream>
#include "graph.h"
#include "wtime.h"
#include "util.h"
#include "countingAlgorithm/sortBased.cuh"
#include "countingAlgorithm/hashBased.cuh"
#include "countingAlgorithm/heapBased.cuh"
#include "countingAlgorithm/mergeBased.cuh"
#include "countingAlgorithm/D_heapBased.cuh"
#include "countingAlgorithm/hashPartition.cuh"
#include "globalPara.h"

#define blocknumber 128

using namespace std;

int BC_subgraph_centric(graph *G)
{

    double startTime, exectionTime;

    long long *D_beginPos;
    int *D_edgeList;

    hipSetDevice(1);
    HRR(hipMalloc(&D_beginPos, sizeof(long long) * (G->uCount + G->vCount + 1)));
    HRR(hipMalloc(&D_edgeList, sizeof(int) * (G->edgeCount)));
    // HRR(hipHostAlloc(&D_beginPos, sizeof(long long) * (G->uCount + G->vCount + 1), hipHostMallocMapped));
    // HRR(hipHostAlloc(&D_edgeList, sizeof(int) * (G->edgeCount), hipHostMallocMapped));
    startTime = wtime();
    HRR(hipMemcpy(D_beginPos, G->beginPos, sizeof(long long) * (G->uCount + G->vCount + 1), hipMemcpyHostToDevice));
    HRR(hipMemcpy(D_edgeList, G->edgeList, sizeof(int) * (G->edgeCount), hipMemcpyHostToDevice));
    exectionTime = wtime() - startTime;
    cout << "load graph elapsed time: " << exectionTime << endl;
    int num_frT = 6, num_srT = 2;
    int *Sorted_List;
    int *host_list;
    HRR(hipMallocManaged((void **)&Sorted_List, sizeof(int) * (G->edgeCount * 2)));
    int total_size = sizeof(int) * (G->edgeCount * 2);
    int *perVertexCount;
    // int * perVertexCount=new int[G->uCount+G->vCount+1];
    HRR(hipMallocManaged((void **)&perVertexCount, sizeof(int) * (G->uCount + G->vCount + 1)));
    int *perVertexCount2;
    // int * perVertexCount=new int[G->uCount+G->vCount+1];
    HRR(hipMallocManaged((void **)&perVertexCount2, sizeof(int) * (G->uCount + G->vCount + 1)));

    unsigned long long *globalCount;
    HRR(hipMallocManaged(&globalCount, sizeof(unsigned long long)));
    *globalCount = 0;
    int *nextVertex;
    HRR(hipMallocManaged(&nextVertex, sizeof(int)));
    int *hashTable;
    HRR(hipMalloc(&hashTable, sizeof(int) * (G->uCount + G->vCount) * blocknumber));

    // startTime = wtime();
    // HRR(hipMemcpy(D_beginPos, G->beginPos, sizeof(long long) * (G->uCount + G->vCount + 1), hipMemcpyHostToDevice));
    // HRR(hipMemcpy(D_edgeList, G->edgeList, sizeof(int) * (G->edgeCount), hipMemcpyHostToDevice));
    // exectionTime = wtime() - startTime;
    // cout << "load graph elapsed time: " << exectionTime << endl;

    if (1)
    {

        long long *D_beginPos_first;
        int *D_edgeList_first;
        long long *D_beginPos_second;
        int *D_edgeList_second;

        HRR(hipMalloc(&D_beginPos_second, sizeof(long long) * (G->uCount + G->vCount + 1)));
        HRR(hipMalloc(&D_edgeList_second, sizeof(int) * (G->subEdgeListSecond[0].size())));

        startTime = wtime();
        double transferTime = 0, computeTime = 0;
        for (int i = 0; i < G->partitionNum; i++)
        {
            startTime = wtime();
            HRR(hipMemcpy(D_beginPos, G->beginPos, sizeof(long long) * (G->uCount + G->vCount + 1), hipMemcpyHostToDevice));
            HRR(hipMemcpy(D_edgeList, G->edgeList, sizeof(int) * (G->edgeCount), hipMemcpyHostToDevice));
            HRR(hipMemcpy(D_beginPos_second, &(G->subBeginPosSecond[i][0]), sizeof(long long) * (G->subBeginPosSecond[i].size()), hipMemcpyHostToDevice));
            HRR(hipMemcpy(D_edgeList_second, &(G->subEdgeListSecond[i][0]), sizeof(int) * (G->subEdgeListSecond[i].size()), hipMemcpyHostToDevice));
            *globalCount = 0;
            transferTime += wtime() - startTime;
            startTime = wtime();
            hashPartition<<<blocknumber, 1024>>>(D_beginPos, D_edgeList, D_beginPos_second, D_edgeList_second, globalCount, perVertexCount, hashTable, 0, G->uCount + G->vCount, G->length, G->partitionNum);
            HRR(hipDeviceSynchronize());
            computeTime += wtime() - startTime;
            cout << *globalCount << endl;
            // cout << G->uCount + G->vCount << endl;
        }
        exectionTime = wtime() - startTime;
        cout << transferTime << ' ' << computeTime << endl;
        // cout << *globalCount << ' ' << exectionTime << endl;
    }

    *globalCount = 0;
    startTime = wtime();
    *nextVertex = blocknumber;
    hashBasedButterflyCounting<<<blocknumber, 1024>>>(D_beginPos, D_edgeList, G->uCount, G->vCount, globalCount, perVertexCount, hashTable, 0, G->breakVertex32, nextVertex);
    HRR(hipDeviceSynchronize());
    exectionTime = wtime() - startTime;
    cout << *globalCount << ' ' << exectionTime << endl;
    cout << "run degree<32 with merge： vertex num: " << G->uCount + G->vCount - G->breakVertex32 << endl;
    startTime = wtime();
    mergeBasedButterflyCounting<<<1024, blockSize>>>(D_beginPos, D_edgeList, G->uCount, G->vCount, globalCount, perVertexCount, hashTable, G->breakVertex32, G->uCount + G->vCount);
    HRR(hipDeviceSynchronize());
    exectionTime = wtime() - startTime;
    cout << *globalCount << ' ' << exectionTime << endl;
    *globalCount = 0;

    // if (0)
    // {
    //     //for test
    //     if (0) //hash>10
    //     {
    //         cout << "run all vertex with hash" << endl;
    //         startTime = wtime();
    //         hashBasedButterflyCounting<<<blocknumber, 1024>>>(D_beginPos, D_edgeList, G->uCount, G->vCount, globalCount, perVertexCount, hashTable, 0, G->uCount + G->vCount);
    //         HRR(hipDeviceSynchronize());
    //         exectionTime = wtime() - startTime;
    //         cout << *globalCount << ' ' << exectionTime << endl;
    //         *globalCount = 0;
    //     }

    //     if (0) //heap<10
    //     {
    //         cout << "run degree<10 with heap： vertex num: " << G->uCount + G->vCount - G->breakVertex10 << endl;
    //         startTime = wtime();
    //         heapBasedButterflyCounting<<<256, 256>>>(D_beginPos, D_edgeList, G->uCount, G->vCount, globalCount, perVertexCount, hashTable, G->breakVertex10, G->uCount + G->vCount);
    //         HRR(hipDeviceSynchronize());
    //         exectionTime = wtime() - startTime;
    //         cout << *globalCount << ' ' << exectionTime << endl;
    //         *globalCount = 0;
    //     }
    //     if (0) //heap<10
    //     {
    //         cout << "run degree<10 with heap on warp： vertex num: " << G->uCount + G->vCount - G->breakVertex10 << endl;
    //         startTime = wtime();
    //         heapBasedButterflyCounting_byWarp<<<512, 1024>>>(D_beginPos, D_edgeList, G->uCount, G->vCount, globalCount, perVertexCount, hashTable, G->breakVertex10, G->uCount + G->vCount);
    //         HRR(hipDeviceSynchronize());
    //         exectionTime = wtime() - startTime;
    //         cout << *globalCount << ' ' << exectionTime << endl;
    //         *globalCount = 0;
    //     }

    //     if (1) //merge<32
    //     {
    //         cout << "run degree<32 with merge： vertex num: " << G->uCount + G->vCount - G->breakVertex32 << endl;
    //         startTime = wtime();
    //         mergeBasedButterflyCounting<<<1024, blockSize>>>(D_beginPos, D_edgeList, G->uCount, G->vCount, globalCount, perVertexCount, hashTable, G->breakVertex32, G->uCount + G->vCount);
    //         HRR(hipDeviceSynchronize());
    //         exectionTime = wtime() - startTime;
    //         cout << *globalCount << ' ' << exectionTime << endl;
    //         *globalCount = 0;
    //     }
    //     if (1) //merge<10
    //     {
    //         cout << "run degree<10 with merge： vertex num: " << G->uCount + G->vCount - G->breakVertex10 << endl;
    //         startTime = wtime();
    //         mergeBasedButterflyCounting<<<1024, blockSize>>>(D_beginPos, D_edgeList, G->uCount, G->vCount, globalCount, perVertexCount, hashTable, G->breakVertex10, G->uCount + G->vCount);
    //         HRR(hipDeviceSynchronize());
    //         exectionTime = wtime() - startTime;
    //         cout << *globalCount << ' ' << exectionTime << endl;
    //         *globalCount = 0;
    //     }

    //     if (1) //hash<32
    //     {
    //         cout << "run degree<32 with hash" << endl;
    //         startTime = wtime();
    //         hashBasedButterflyCounting<<<blocknumber, 1024>>>(D_beginPos, D_edgeList, G->uCount, G->vCount, globalCount, perVertexCount, hashTable, G->breakVertex32, G->uCount + G->vCount);
    //         HRR(hipDeviceSynchronize());
    //         exectionTime = wtime() - startTime;
    //         cout << *globalCount << ' ' << exectionTime << endl;
    //         *globalCount = 0;
    //     }
    //     if (1) //hash<10
    //     {
    //         cout << "run degree<10 with hash" << endl;
    //         startTime = wtime();
    //         hashBasedButterflyCounting<<<blocknumber, 1024>>>(D_beginPos, D_edgeList, G->uCount, G->vCount, globalCount, perVertexCount, hashTable, G->breakVertex10, G->uCount + G->vCount);
    //         HRR(hipDeviceSynchronize());
    //         exectionTime = wtime() - startTime;
    //         cout << *globalCount << ' ' << exectionTime << endl;
    //         *globalCount = 0;
    //     }

    //     if (0) //for debug
    //     {
    //         for (int i = 71869 - 2; i <= 71869; i++)
    //         {
    //             *globalCount = 0;
    //             startTime = wtime();
    //             mergeBasedButterflyCounting<<<1, blockSize>>>(D_beginPos, D_edgeList, G->uCount, G->vCount, globalCount, perVertexCount, hashTable, i, i + 1);
    //             HRR(hipDeviceSynchronize());
    //             exectionTime = wtime() - startTime;
    //             int res1 = *globalCount;
    //             *globalCount = 0;
    //             startTime = wtime();
    //             heapBasedButterflyCounting<<<blocknumber, 1024>>>(D_beginPos, D_edgeList, G->uCount, G->vCount, globalCount, perVertexCount, hashTable, i, i + 1);
    //             HRR(hipDeviceSynchronize());
    //             exectionTime = wtime() - startTime;
    //             int res2 = *globalCount;
    //             if (res1 != res2)
    //                 printf("%d,%d,%d\n", res1, res2, i);
    //             *globalCount = 0;
    //         }
    //     }
    //     if (0) //heap<10
    //     {
    //         cout << "run degree<10 with D_heap" << endl;
    //         startTime = wtime();
    //         D_heapBasedButterflyCounting<<<blocknumber, 128>>>(D_beginPos, D_edgeList, Sorted_List, total_size, num_frT, num_srT, G->uCount, G->vCount, globalCount, perVertexCount, hashTable, G->breakVertex10, G->uCount + G->vCount);
    //         HRR(hipDeviceSynchronize());
    //         exectionTime = wtime() - startTime;
    //         cout << *globalCount << ' ' << exectionTime << endl;
    //         *globalCount = 0;
    //     }
    //     if (0) //heap<100
    //     {
    //         cout << "run all vertex with D_heap" << endl;
    //         startTime = wtime();
    //         D_heapBasedButterflyCounting<<<blocknumber, 128>>>(D_beginPos, D_edgeList, Sorted_List, total_size, num_frT, num_srT, G->uCount, G->vCount, globalCount, perVertexCount, hashTable, 0, G->uCount + G->vCount);
    //         HRR(hipDeviceSynchronize());
    //         exectionTime = wtime() - startTime;
    //         cout << *globalCount << ' ' << exectionTime << endl;
    //         *globalCount = 0;
    //     }
    //     if (0) //hash>32
    //     {
    //         startTime = wtime();
    //         hashBasedButterflyCounting<<<blocknumber, 1024>>>(D_beginPos, D_edgeList, G->uCount, G->vCount, globalCount, perVertexCount, hashTable, 0, G->breakVertex32);
    //         HRR(hipDeviceSynchronize());
    //         exectionTime = wtime() - startTime;
    //         cout << ' ' << exectionTime;
    //     }
    //     if (0) //hash<32
    //     {
    //         startTime = wtime();
    //         hashBasedButterflyCounting<<<blocknumber, 1024>>>(D_beginPos, D_edgeList, G->uCount, G->vCount, globalCount, perVertexCount, hashTable, G->breakVertex32, G->uCount + G->vCount);
    //         HRR(hipDeviceSynchronize());
    //         exectionTime = wtime() - startTime;
    //         cout << ' ' << exectionTime;
    //     }
    //     if (0) //sort<32
    //     {
    //         startTime = wtime();
    //         sortBasedButterflyCounting<<<blocknumber, 1024>>>(D_beginPos, D_edgeList, G->uCount, G->vCount, globalCount, perVertexCount, hashTable, G->breakVertex32, G->uCount + G->vCount);
    //         HRR(hipDeviceSynchronize());
    //         exectionTime = wtime() - startTime;
    //         cout << ' ' << exectionTime;
    //     }
    //     if (0) //10<hash<32
    //     {
    //         startTime = wtime();
    //         hashBasedButterflyCounting<<<blocknumber, 1024>>>(D_beginPos, D_edgeList, G->uCount, G->vCount, globalCount, perVertexCount, hashTable, G->breakVertex32, G->breakVertex10);
    //         HRR(hipDeviceSynchronize());
    //         exectionTime = wtime() - startTime;
    //         cout << ' ' << exectionTime;
    //     }
    // }

    // if (0) //combined several method
    // {
    //     startTime = wtime();
    //     hashBasedButterflyCounting<<<blocknumber, 1024>>>(D_beginPos, D_edgeList, G->uCount, G->vCount, globalCount, perVertexCount, hashTable, 0, G->breakVertex10);
    //     HRR(hipDeviceSynchronize());
    //     exectionTime = wtime() - startTime;
    //     cout << *globalCount << ' ' << exectionTime << endl;

    //     startTime = wtime();
    //     heapBasedButterflyCounting<<<blocknumber, 128>>>(D_beginPos, D_edgeList, G->uCount, G->vCount, globalCount, perVertexCount, hashTable, G->breakVertex10, G->uCount + G->vCount);
    //     HRR(hipDeviceSynchronize());
    //     exectionTime = wtime() - startTime;
    //     cout << *globalCount << ' ' << exectionTime << endl;
    //     *globalCount = 0;
    // }

    // *globalCount=0;
    // startTime=wtime();
    // sortBasedButterflyCounting<<<blocknumber,1024>>>(D_beginPos,D_edgeList,G->uCount,G->vCount,globalCount,perVertexCount2,hashTable,G->breakVertex32,G->vertexCount);
    // HRR(hipDeviceSynchronize());
    // exectionTime=wtime()-startTime;
    // cout<<*globalCount<<' '<<exectionTime;
    // for (int i=G->breakVertex32;i<G->breakVertex10;i++)
    //     if (perVertexCount[i]!=perVertexCount2[i])
    //         cout<<i<<endl;
    cout << endl;

    // HRR(hipMemcpy((void **)&host_list,(void **)&Sorted_List,sizeof(int)*(G->edgeCount), hipMemcpyDeviceToHost));

    HRR(hipFree(D_beginPos));
    HRR(hipFree(D_edgeList));
    HRR(hipFree(Sorted_List));

    // delete(perVertexCount);
    return 0;
}

int BC_hashtable_centric(graph *G)
{

    double startTime, exectionTime;

    long long *D_beginPos;
    int *D_edgeList;

    hipSetDevice(1);
    HRR(hipMalloc(&D_beginPos, sizeof(long long) * (G->uCount + G->vCount + 1)));
    HRR(hipMalloc(&D_edgeList, sizeof(int) * (G->edgeCount)));
    startTime = wtime();
    HRR(hipMemcpy(D_beginPos, G->beginPos, sizeof(long long) * (G->uCount + G->vCount + 1), hipMemcpyHostToDevice));
    HRR(hipMemcpy(D_edgeList, G->edgeList, sizeof(int) * (G->edgeCount), hipMemcpyHostToDevice));
    exectionTime = wtime() - startTime;
    cout << "load graph elapsed time: " << exectionTime << endl;
    unsigned long long *globalCount;
    HRR(hipMallocManaged(&globalCount, sizeof(unsigned long long)));
    *globalCount = 0;
    int *nextVertex;
    HRR(hipMallocManaged(&nextVertex, sizeof(int)));
    int *hashTable;
    int vertexNum = (G->uCount + G->vCount) / G->partitionNum + 1;
    HRR(hipMalloc(&hashTable, sizeof(int) * vertexNum * vertexNum));

    long long *D_beginPos_first;
    int *D_edgeList_first;
    long long *D_beginPos_second;
    int *D_edgeList_second;

    HRR(hipMalloc(&D_beginPos_second, sizeof(long long) * (G->uCount + G->vCount + 1)));
    HRR(hipMalloc(&D_edgeList_second, sizeof(int) * (G->subEdgeListSecond[0].size())));

    startTime = wtime();
    double transferTime = 0, computeTime = 0;
    for (int i = 0; i < G->partitionNum; i++)
    {
        startTime = wtime();
        HRR(hipMemcpy(D_beginPos_second, &(G->subBeginPosSecond[i][0]), sizeof(long long) * (G->subBeginPosSecond[i].size()), hipMemcpyHostToDevice));
        HRR(hipMemcpy(D_edgeList_second, &(G->subEdgeListSecond[i][0]), sizeof(int) * (G->subEdgeListSecond[i].size()), hipMemcpyHostToDevice));
        *globalCount = 0;
        transferTime += wtime() - startTime;
        startTime = wtime();
        hashPartition<<<blocknumber, 1024>>>(D_beginPos, D_edgeList, D_beginPos_second, D_edgeList_second, globalCount, perVertexCount, hashTable, 0, G->uCount + G->vCount, G->length, G->partitionNum);
        HRR(hipDeviceSynchronize());
        computeTime += wtime() - startTime;
        cout << *globalCount << endl;
        // cout << G->uCount + G->vCount << endl;
    }
    exectionTime = wtime() - startTime;
    cout << transferTime << ' ' << computeTime << endl;
    // cout << *globalCount << ' ' << exectionTime << endl;

    HRR(hipFree(D_beginPos));
    HRR(hipFree(D_edgeList));

    return 0;
}
