#include "hip/hip_runtime.h"
#include <iostream>
#include "graph.h"
#include "wtime.h"
#include "util.h"
#include <hipcub/hipcub.hpp>
#include <cub/util_type.cuh>
#include "countingAlgorithm/sortBased.cuh"
#include "countingAlgorithm/hashBased.cuh"
#include "countingAlgorithm/heapBased.cuh"
#include "countingAlgorithm/D_heapBased.cuh"

#define blocknumber 128

using namespace std;
// using namespace hipcub;

template <int BLOCK_THREADS, int ITEMS_PER_THREAD>
__global__ void neighborSorting(int* d_in)
{
    // int vertex=blockIdx.x;
    typedef hipcub::BlockLoad<int, BLOCK_THREADS, ITEMS_PER_THREAD, hipcub::BLOCK_LOAD_TRANSPOSE> BlockLoadT;    
    typedef hipcub::BlockStore<int, BLOCK_THREADS, ITEMS_PER_THREAD, hipcub::BLOCK_STORE_TRANSPOSE> BlockStoreT;
    typedef hipcub::BlockRadixSort<int, BLOCK_THREADS, ITEMS_PER_THREAD> BlockRadixSortT;
    __shared__ union {
        typename BlockLoadT::TempStorage       load; 
        typename BlockStoreT::TempStorage      store; 
        typename BlockRadixSortT::TempStorage  sort;
    } temp_storage; 

    int thread_keys[ITEMS_PER_THREAD];
    // int *p;
    // p=thread_keys;
    int block_offset = blockIdx.x * (BLOCK_THREADS * ITEMS_PER_THREAD);      
    BlockLoadT(temp_storage.load).Load(d_in + block_offset, thread_keys);

    // for (int i=threadIdx.x;i<ITEMS_PER_THREAD*BLOCK_THREADS;i+=BLOCK_THREADS)
    //     thread_keys[i/BLOCK_THREADS]=d_in[i];
    __syncthreads();    // Barrier for smem reuse
    // if (threadIdx.x==0)
    // {
    //     for (int i=0;i<ITEMS_PER_THREAD;i++)
    //     printf("%d ",thread_keys[i]);
    // }
    // Collectively sort the keys
    BlockRadixSortT(temp_storage.sort).Sort(thread_keys);
    // if (threadIdx.x==0)
    // {
    //     for (int i=0;i<ITEMS_PER_THREAD;i++)
    //     printf("%d ",thread_keys[i]);
    // }
    __syncthreads();    // Barrier for smem reuse
    // Store the sorted segment 
    BlockStoreT(temp_storage.store).Store(d_in + block_offset, thread_keys);

}

void sort_test()
{
    
    double startTime,exectionTime;
    int *d_in,*in;
    int num_blocks=1;
    const int num_per_thread=8;
    const int num_thread=1024;
    int N=num_thread*num_per_thread*num_blocks;
    cout<<N<<endl;
    in=new int[N];
    for (int i=0;i<N;i++)
    {
        in[i]=N-i;
    }
    HRR(hipMalloc((void **)&d_in, N*sizeof(int)));
    HRR(hipMemcpy(d_in,in,sizeof(int)*N, hipMemcpyHostToDevice));

    neighborSorting<num_thread, num_per_thread><<<num_blocks, num_thread>>>(d_in); 
    startTime=wtime();
    HRR(hipDeviceSynchronize());
    exectionTime=wtime()-startTime;
    
    HRR(hipMemcpy(in,d_in,sizeof(int)*N, hipMemcpyDeviceToHost));
    cout<<in[100]<<' '<<exectionTime<<endl;
}


int BC(graph* G)
{
    
    double startTime,exectionTime;

    long long* D_beginPos;
    int* D_edgeList;
    HRR(hipMalloc((void **) &D_beginPos,sizeof(long long)*(G->uCount+G->vCount+1)));
    HRR(hipMalloc((void **) &D_edgeList,sizeof(int)*(G->edgeCount)));
    HRR(hipMemcpy(D_beginPos,G->beginPos,sizeof(long long)*(G->uCount+G->vCount+1), hipMemcpyHostToDevice));
    HRR(hipMemcpy(D_edgeList,G->edgeList,sizeof(int)*(G->edgeCount), hipMemcpyHostToDevice));
    int num_frT=6,num_srT=2;
    int* Sorted_List;
    int* host_list;
    HRR(hipMallocManaged((void **) &Sorted_List,sizeof(int)*(G->edgeCount*2)));
    int total_size=sizeof(int)*(G->edgeCount*2);
    int * perVertexCount;
    // int * perVertexCount=new int[G->uCount+G->vCount+1];
    HRR(hipMallocManaged((void **) &perVertexCount,sizeof(int)*(G->uCount+G->vCount+1)));
    int * perVertexCount2;
    // int * perVertexCount=new int[G->uCount+G->vCount+1];
    HRR(hipMallocManaged((void **) &perVertexCount2,sizeof(int)*(G->uCount+G->vCount+1)));

    unsigned long long *globalCount;
    HRR(hipMallocManaged(&globalCount, sizeof(unsigned long long)));
    *globalCount=0;
    int *hashTable;
    HRR(hipMalloc(&hashTable, sizeof(int)*(G->uCount+G->vCount)*blocknumber));

        
    if (1)
    {
        //for test
        if (1) //hash>10
        {
        cout<<"run all vertex with hash"<<endl;
            startTime=wtime();
            hashBasedButterflyCounting<<<blocknumber,1024>>>(D_beginPos,D_edgeList,G->uCount,G->vCount,globalCount,perVertexCount,hashTable,0,G->uCount+G->vCount);
            HRR(hipDeviceSynchronize());
            exectionTime=wtime()-startTime;
            cout<<*globalCount<<' '<<exectionTime<<endl;
            *globalCount=0;
        }
        
        if (1) //hash<10
        {
        cout<<"run degree<10 with hash"<<endl;
            startTime=wtime();
            hashBasedButterflyCounting<<<blocknumber,1024>>>(D_beginPos,D_edgeList,G->uCount,G->vCount,globalCount,perVertexCount,hashTable,G->breakVertex10,G->uCount+G->vCount);
            HRR(hipDeviceSynchronize());
            exectionTime=wtime()-startTime;
            cout<<*globalCount<<' '<<exectionTime<<endl;
            *globalCount=0;
        }
        if (1) //heap<10
        {
        cout<<"run degree<10 with heap： vertex num: "<<G->uCount+G->vCount-G->breakVertex10<<endl;
            startTime=wtime();
            heapBasedButterflyCounting<<<blocknumber,128>>>(D_beginPos,D_edgeList,G->uCount,G->vCount,globalCount,perVertexCount,hashTable,G->breakVertex10,G->uCount+G->vCount);
            HRR(hipDeviceSynchronize());
            exectionTime=wtime()-startTime;
            cout<<*globalCount<<' '<<exectionTime<<endl;
            *globalCount=0;
        }
        if (1) //heap<10
        {
        cout<<"run degree<10 with D_heap"<<endl;
            startTime=wtime();
            D_heapBasedButterflyCounting<<<blocknumber,128>>>(D_beginPos,D_edgeList,Sorted_List,total_size,num_frT,num_srT,G->uCount,G->vCount,globalCount,perVertexCount,hashTable,G->breakVertex10,G->uCount+G->vCount);
            HRR(hipDeviceSynchronize());
            exectionTime=wtime()-startTime;
            cout<<*globalCount<<' '<<exectionTime<<endl;
            *globalCount=0;
        }
        if (1) //heap<100
        {
        cout<<"run all vertex with D_heap"<<endl;
            startTime=wtime();
            D_heapBasedButterflyCounting<<<blocknumber,128>>>(D_beginPos,D_edgeList,Sorted_List,total_size,num_frT,num_srT,G->uCount,G->vCount,globalCount,perVertexCount,hashTable,0,G->uCount+G->vCount);
            HRR(hipDeviceSynchronize());
            exectionTime=wtime()-startTime;
            cout<<*globalCount<<' '<<exectionTime<<endl;
            *globalCount=0;
        }
        if (0) //hash>32
        {
            startTime=wtime();
            hashBasedButterflyCounting<<<blocknumber,1024>>>(D_beginPos,D_edgeList,G->uCount,G->vCount,globalCount,perVertexCount,hashTable,0,G->breakVertex32);
            HRR(hipDeviceSynchronize());
            exectionTime=wtime()-startTime;
            cout<<' '<<exectionTime;
        }
        if (0) //hash<32
        {
            startTime=wtime();
            hashBasedButterflyCounting<<<blocknumber,1024>>>(D_beginPos,D_edgeList,G->uCount,G->vCount,globalCount,perVertexCount,hashTable,G->breakVertex32,G->uCount+G->vCount);
            HRR(hipDeviceSynchronize());
            exectionTime=wtime()-startTime;
            cout<<' '<<exectionTime;
        }
        if (0) //sort<32
        {
            startTime=wtime();
            sortBasedButterflyCounting<<<blocknumber,1024>>>(D_beginPos,D_edgeList,G->uCount,G->vCount,globalCount,perVertexCount,hashTable,G->breakVertex32,G->uCount+G->vCount);
            HRR(hipDeviceSynchronize());
            exectionTime=wtime()-startTime;
            cout<<' '<<exectionTime;
        }
        if (0) //10<hash<32
        {
            startTime=wtime();
            hashBasedButterflyCounting<<<blocknumber,1024>>>(D_beginPos,D_edgeList,G->uCount,G->vCount,globalCount,perVertexCount,hashTable,G->breakVertex32,G->breakVertex10);
            HRR(hipDeviceSynchronize());
            exectionTime=wtime()-startTime;
            cout<<' '<<exectionTime;
        }
    }

/*
    startTime=wtime();
    hashBasedButterflyCounting<<<blocknumber,1024>>>(D_beginPos,D_edgeList,G->uCount,G->vCount,globalCount,perVertexCount,hashTable,0,G->breakVertex32);
    HRR(hipDeviceSynchronize());
    exectionTime=wtime()-startTime;
    cout<<*globalCount<<' '<<exectionTime<<endl;

    
    // *globalCount=0;
    startTime=wtime();
    sortBasedButterflyCounting<<<blocknumber,1024>>>(D_beginPos,D_edgeList,G->uCount,G->vCount,globalCount,perVertexCount2,hashTable,G->breakVertex32,G->vertexCount);
    HRR(hipDeviceSynchronize());
    exectionTime=wtime()-startTime;
    cout<<*globalCount<<' '<<exectionTime;
    // for (int i=G->breakVertex32;i<G->breakVertex10;i++)
    //     if (perVertexCount[i]!=perVertexCount2[i])
    //         cout<<i<<endl;
    cout<<endl;
    */
   // HRR(hipMemcpy((void **)&host_list,(void **)&Sorted_List,sizeof(int)*(G->edgeCount), hipMemcpyDeviceToHost));
    
    HRR(hipFree(D_beginPos));
    HRR(hipFree(D_edgeList));
    HRR(hipFree(Sorted_List));
    
    // delete(perVertexCount);
    return 0;
}

