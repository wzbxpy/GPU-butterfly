#include "hip/hip_runtime.h"
#include <iostream>
#include "graph.h"
#include "wtime.h"
#include "util.h"
#include "countingAlgorithm/sortBased.cuh"
#include "countingAlgorithm/hashBased.cuh"
#include "countingAlgorithm/heapBased.cuh"
#include "countingAlgorithm/mergeBased.cuh"
#include "countingAlgorithm/D_heapBased.cuh"
#include "countingAlgorithm/hashPartition.cuh"
#include "countingAlgorithm/hashCentric.cuh"
#include "globalPara.h"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/memcpy_async.h>
#include <cooperative_groups/reduce.h>
// #include <cooperative_groups/scan.h>
#define dev 1

using namespace std;
using namespace cooperative_groups;

__global__ void test(unsigned long long *count)
{
    if (this_thread_block().thread_rank() < 100)
        atomicAdd(count, 1);

    this_thread_block().sync();
    // __syncthreads();
    int x = *count;
    // __syncthreads();
    this_thread_block().sync();
    *count = 0;
    // this_thread_block().sync();
    __syncthreads();
    atomicAdd(count, x);
}

template <class T>
int initializeCudaPara(int deviceId, int numThreads, T func)
{
    hipSetDevice(deviceId);
    int numBlocksPerSm = 0;
    // Number of threads my_kernel will be launched with
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, deviceId);
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm, func, numThreads, 0);
    cout << deviceProp.multiProcessorCount << "  " << numBlocksPerSm << endl;
    int numBlocks = deviceProp.multiProcessorCount * numBlocksPerSm;
    return numBlocks;
}

int BC_subgraph_centric(graph *G)
{
    double startTime, exectionTime;

    int numThreads = 1024;
    int numBlocks = initializeCudaPara(dev, numThreads, hashCentric);
    numBlocks = 128;

    long long *D_beginPos;
    int *D_edgeList;

    HRR(hipMalloc(&D_beginPos, sizeof(long long) * (G->uCount + G->vCount + 1)));
    HRR(hipMalloc(&D_edgeList, sizeof(int) * (G->edgeCount)));
    startTime = wtime();
    HRR(hipMemcpy(D_beginPos, G->beginPos, sizeof(long long) * (G->uCount + G->vCount + 1), hipMemcpyHostToDevice));
    HRR(hipMemcpy(D_edgeList, G->edgeList, sizeof(int) * (G->edgeCount), hipMemcpyHostToDevice));
    exectionTime = wtime() - startTime;
    cout << "load graph elapsed time: " << exectionTime << endl;
    int num_frT = 6, num_srT = 2;
    int *Sorted_List;
    int *host_list;
    HRR(hipMallocManaged((void **)&Sorted_List, sizeof(int) * (G->edgeCount * 2)));
    int total_size = sizeof(int) * (G->edgeCount * 2);
    int *perVertexCount;
    // int * perVertexCount=new int[G->uCount+G->vCount+1];
    HRR(hipMallocManaged((void **)&perVertexCount, sizeof(int) * (G->uCount + G->vCount + 1)));

    unsigned long long *globalCount;
    HRR(hipMallocManaged(&globalCount, sizeof(unsigned long long)));
    *globalCount = 0;
    int *hashTable;
    HRR(hipMalloc(&hashTable, sizeof(int) * (G->uCount + G->vCount) * numBlocks));

    test<<<1, 1024>>>(globalCount);
    HRR(hipDeviceSynchronize());
    cout << "here " << *globalCount << endl;

    if (0)
    {

        long long *D_beginPos_first;
        int *D_edgeList_first;
        long long *D_beginPos_second;
        int *D_edgeList_second;

        HRR(hipMalloc(&D_beginPos_first, sizeof(long long) * G->subBeginPosFirst[0].size()));
        HRR(hipMalloc(&D_edgeList_first, sizeof(int) * (G->subEdgeListFirst[0].size())));
        HRR(hipMalloc(&D_beginPos_second, sizeof(long long) * (G->uCount + G->vCount + 1)));
        HRR(hipMalloc(&D_edgeList_second, sizeof(int) * (G->subEdgeListSecond[0].size())));

        startTime = wtime();
        double transferTime = 0, computeTime = 0;
        for (int j = 0; j < G->partitionNum; j++)
        // for (int j = 0; j < 1; j++)
        {

            HRR(hipMemcpy(D_beginPos_first, &(G->subBeginPosFirst[j][0]), sizeof(long long) * (G->subBeginPosFirst[j].size()), hipMemcpyHostToDevice));
            HRR(hipMemcpy(D_edgeList_first, &(G->subEdgeListFirst[j][0]), sizeof(int) * (G->subEdgeListFirst[j].size()), hipMemcpyHostToDevice));
            for (int i = 0; i < G->partitionNum; i++)
            {
                // cout << i << ' ' << j << endl;
                startTime = wtime();
                HRR(hipMemcpy(D_beginPos_second, &(G->subBeginPosSecond[i][0]), sizeof(long long) * (G->subBeginPosSecond[i].size()), hipMemcpyHostToDevice));
                HRR(hipMemcpy(D_edgeList_second, &(G->subEdgeListSecond[i][0]), sizeof(int) * (G->subEdgeListSecond[i].size()), hipMemcpyHostToDevice));
                // *globalCount = 0;
                transferTime += wtime() - startTime;
                startTime = wtime();
                hashPartition<<<numBlocks, numThreads>>>(D_beginPos_first, D_edgeList_first, D_beginPos_second, D_edgeList_second, globalCount, perVertexCount, hashTable, 0, G->subBeginPosFirst[j].size() - 1, G->length, G->partitionNum, j);
                HRR(hipDeviceSynchronize());
                computeTime += wtime() - startTime;
                // cout << G->uCount + G->vCount << endl;
            }
        }
        cout << *globalCount << endl;
        exectionTime = wtime() - startTime;
        cout << transferTime << ' ' << computeTime << endl;
        // cout << *globalCount << ' ' << exectionTime << endl;
    }

    cout << endl;

    // HRR(hipMemcpy((void **)&host_list,(void **)&Sorted_List,sizeof(int)*(G->edgeCount), hipMemcpyDeviceToHost));

    HRR(hipFree(D_beginPos));
    HRR(hipFree(D_edgeList));
    HRR(hipFree(Sorted_List));

    // delete(perVertexCount);
    return 0;
}

int BC_hashtable_centric(graph *G)
{

    double startTime, exectionTime;

    long long *D_beginPos;
    int *D_edgeList;
    int numThreads = 1024;
    int numBlocks = initializeCudaPara(dev, numThreads, hashCentric);

    HRR(hipMalloc(&D_beginPos, sizeof(long long) * (G->uCount + G->vCount + 1)));
    HRR(hipMalloc(&D_edgeList, sizeof(int) * (G->edgeCount)));
    startTime = wtime();
    HRR(hipMemcpy(D_beginPos, G->beginPos, sizeof(long long) * (G->uCount + G->vCount + 1), hipMemcpyHostToDevice));
    HRR(hipMemcpy(D_edgeList, G->edgeList, sizeof(int) * (G->edgeCount), hipMemcpyHostToDevice));
    exectionTime = wtime() - startTime;
    cout << "load graph elapsed time: " << exectionTime << endl;
    unsigned long long *globalCount;
    HRR(hipMallocManaged(&globalCount, sizeof(unsigned long long)));
    *globalCount = 0;
    int *nextVertex;
    HRR(hipMallocManaged(&nextVertex, sizeof(int)));
    int *hashTable;
    HRR(hipMalloc(&hashTable, sizeof(int) * G->length * G->length));

    long long *D_beginPos_first;
    int *D_edgeList_first;
    long long *D_beginPos_second;
    int *D_edgeList_second;

    HRR(hipMalloc(&D_beginPos_first, sizeof(long long) * (G->uCount + G->vCount + 1)));
    HRR(hipMalloc(&D_edgeList_first, sizeof(int) * (G->subEdgeListSecond[0].size())));
    HRR(hipMalloc(&D_beginPos_second, sizeof(long long) * (G->uCount + G->vCount + 1)));
    HRR(hipMalloc(&D_edgeList_second, sizeof(int) * (G->subEdgeListSecond[0].size())));

    cout << G->vertexCount / 100 << " number of vetrex" << endl;
    *globalCount = 0;
    startTime = wtime();
    double transferTime = 0, computeTime = 0;
    for (int i = 0; i < G->partitionNum; i++)
    {
        HRR(hipMemcpy(D_beginPos_first, &(G->subBeginPosSecond[i][0]), sizeof(long long) * (G->subBeginPosSecond[i].size()), hipMemcpyHostToDevice));
        HRR(hipMemcpy(D_edgeList_first, &(G->subEdgeListSecond[i][0]), sizeof(int) * (G->subEdgeListSecond[i].size()), hipMemcpyHostToDevice));
        for (int j = 0; j < G->partitionNum; j++)
        {
            // cout << i << ' ' << j << endl;
            startTime = wtime();
            HRR(hipMemcpy(D_beginPos_second, &(G->subBeginPosSecond[j][0]), sizeof(long long) * (G->subBeginPosSecond[j].size()), hipMemcpyHostToDevice));
            HRR(hipMemcpy(D_edgeList_second, &(G->subEdgeListSecond[j][0]), sizeof(int) * (G->subEdgeListSecond[j].size()), hipMemcpyHostToDevice));
            // *globalCount = 0;
            transferTime += wtime() - startTime;
            startTime = wtime();
            // clearHashTable<<<G->length, 1024>>>(hashTable, G->length);
            // HRR(hipDeviceSynchronize());
            int startVertex = 0;
            void *kernelArgs[] = {&D_beginPos_first, &D_edgeList_first, &D_beginPos_second, &D_edgeList_second, &globalCount, &hashTable, &startVertex, &G->vertexCount, &G->length, &G->partitionNum};
            hipLaunchCooperativeKernel((void *)hashCentric, numBlocks, numThreads, kernelArgs);
            // hashCentric<<<numBlocks, 1024>>>(D_beginPos_first, D_edgeList_first, D_beginPos_second, D_edgeList_second, globalCount, hashTable, 0, G->uCount + G->vCount, G->length, G->partitionNum);
            HRR(hipDeviceSynchronize());
            computeTime += wtime() - startTime;
            // cout << *globalCount << endl;
            // cout << G->uCount + G->vCount << endl;
        }
    }
    cout << *globalCount << endl;
    exectionTime = wtime() - startTime;
    cout << transferTime << ' ' << computeTime << endl;
    // cout << *globalCount << ' ' << exectionTime << endl;

    HRR(hipFree(D_beginPos));
    HRR(hipFree(D_edgeList));

    return 0;
}

// *globalCount = 0;
// startTime = wtime();
// *nextVertex = numBlocks;
// hashBasedButterflyCounting<<<numBlocks, 1024>>>(D_beginPos, D_edgeList, G->uCount, G->vCount, globalCount, perVertexCount, hashTable, 0, G->breakVertex32, nextVertex);
// HRR(hipDeviceSynchronize());
// exectionTime = wtime() - startTime;
// cout << *globalCount << ' ' << exectionTime << endl;
// cout << "run degree<32 with merge： vertex num: " << G->uCount + G->vCount - G->breakVertex32 << endl;
// startTime = wtime();
// mergeBasedButterflyCounting<<<1024, blockSize>>>(D_beginPos, D_edgeList, G->uCount, G->vCount, globalCount, perVertexCount, hashTable, G->breakVertex32, G->uCount + G->vCount);
// HRR(hipDeviceSynchronize());
// exectionTime = wtime() - startTime;
// cout << *globalCount << ' ' << exectionTime << endl;
// *globalCount = 0;

// if (0)
// {
//     //for test
//     if (0) //hash>10
//     {
//         cout << "run all vertex with hash" << endl;
//         startTime = wtime();
//         hashBasedButterflyCounting<<<numBlocks, 1024>>>(D_beginPos, D_edgeList, G->uCount, G->vCount, globalCount, perVertexCount, hashTable, 0, G->uCount + G->vCount);
//         HRR(hipDeviceSynchronize());
//         exectionTime = wtime() - startTime;
//         cout << *globalCount << ' ' << exectionTime << endl;
//         *globalCount = 0;
//     }

//     if (0) //heap<10
//     {
//         cout << "run degree<10 with heap： vertex num: " << G->uCount + G->vCount - G->breakVertex10 << endl;
//         startTime = wtime();
//         heapBasedButterflyCounting<<<256, 256>>>(D_beginPos, D_edgeList, G->uCount, G->vCount, globalCount, perVertexCount, hashTable, G->breakVertex10, G->uCount + G->vCount);
//         HRR(hipDeviceSynchronize());
//         exectionTime = wtime() - startTime;
//         cout << *globalCount << ' ' << exectionTime << endl;
//         *globalCount = 0;
//     }
//     if (0) //heap<10
//     {
//         cout << "run degree<10 with heap on warp： vertex num: " << G->uCount + G->vCount - G->breakVertex10 << endl;
//         startTime = wtime();
//         heapBasedButterflyCounting_byWarp<<<512, 1024>>>(D_beginPos, D_edgeList, G->uCount, G->vCount, globalCount, perVertexCount, hashTable, G->breakVertex10, G->uCount + G->vCount);
//         HRR(hipDeviceSynchronize());
//         exectionTime = wtime() - startTime;
//         cout << *globalCount << ' ' << exectionTime << endl;
//         *globalCount = 0;
//     }

//     if (1) //merge<32
//     {
//         cout << "run degree<32 with merge： vertex num: " << G->uCount + G->vCount - G->breakVertex32 << endl;
//         startTime = wtime();
//         mergeBasedButterflyCounting<<<1024, blockSize>>>(D_beginPos, D_edgeList, G->uCount, G->vCount, globalCount, perVertexCount, hashTable, G->breakVertex32, G->uCount + G->vCount);
//         HRR(hipDeviceSynchronize());
//         exectionTime = wtime() - startTime;
//         cout << *globalCount << ' ' << exectionTime << endl;
//         *globalCount = 0;
//     }
//     if (1) //merge<10
//     {
//         cout << "run degree<10 with merge： vertex num: " << G->uCount + G->vCount - G->breakVertex10 << endl;
//         startTime = wtime();
//         mergeBasedButterflyCounting<<<1024, blockSize>>>(D_beginPos, D_edgeList, G->uCount, G->vCount, globalCount, perVertexCount, hashTable, G->breakVertex10, G->uCount + G->vCount);
//         HRR(hipDeviceSynchronize());
//         exectionTime = wtime() - startTime;
//         cout << *globalCount << ' ' << exectionTime << endl;
//         *globalCount = 0;
//     }

//     if (1) //hash<32
//     {
//         cout << "run degree<32 with hash" << endl;
//         startTime = wtime();
//         hashBasedButterflyCounting<<<numBlocks, 1024>>>(D_beginPos, D_edgeList, G->uCount, G->vCount, globalCount, perVertexCount, hashTable, G->breakVertex32, G->uCount + G->vCount);
//         HRR(hipDeviceSynchronize());
//         exectionTime = wtime() - startTime;
//         cout << *globalCount << ' ' << exectionTime << endl;
//         *globalCount = 0;
//     }
//     if (1) //hash<10
//     {
//         cout << "run degree<10 with hash" << endl;
//         startTime = wtime();
//         hashBasedButterflyCounting<<<numBlocks, 1024>>>(D_beginPos, D_edgeList, G->uCount, G->vCount, globalCount, perVertexCount, hashTable, G->breakVertex10, G->uCount + G->vCount);
//         HRR(hipDeviceSynchronize());
//         exectionTime = wtime() - startTime;
//         cout << *globalCount << ' ' << exectionTime << endl;
//         *globalCount = 0;
//     }

//     if (0) //for debug
//     {
//         for (int i = 71869 - 2; i <= 71869; i++)
//         {
//             *globalCount = 0;
//             startTime = wtime();
//             mergeBasedButterflyCounting<<<1, blockSize>>>(D_beginPos, D_edgeList, G->uCount, G->vCount, globalCount, perVertexCount, hashTable, i, i + 1);
//             HRR(hipDeviceSynchronize());
//             exectionTime = wtime() - startTime;
//             int res1 = *globalCount;
//             *globalCount = 0;
//             startTime = wtime();
//             heapBasedButterflyCounting<<<numBlocks, 1024>>>(D_beginPos, D_edgeList, G->uCount, G->vCount, globalCount, perVertexCount, hashTable, i, i + 1);
//             HRR(hipDeviceSynchronize());
//             exectionTime = wtime() - startTime;
//             int res2 = *globalCount;
//             if (res1 != res2)
//                 printf("%d,%d,%d\n", res1, res2, i);
//             *globalCount = 0;
//         }
//     }
//     if (0) //heap<10
//     {
//         cout << "run degree<10 with D_heap" << endl;
//         startTime = wtime();
//         D_heapBasedButterflyCounting<<<numBlocks, 128>>>(D_beginPos, D_edgeList, Sorted_List, total_size, num_frT, num_srT, G->uCount, G->vCount, globalCount, perVertexCount, hashTable, G->breakVertex10, G->uCount + G->vCount);
//         HRR(hipDeviceSynchronize());
//         exectionTime = wtime() - startTime;
//         cout << *globalCount << ' ' << exectionTime << endl;
//         *globalCount = 0;
//     }
//     if (0) //heap<100
//     {
//         cout << "run all vertex with D_heap" << endl;
//         startTime = wtime();
//         D_heapBasedButterflyCounting<<<numBlocks, 128>>>(D_beginPos, D_edgeList, Sorted_List, total_size, num_frT, num_srT, G->uCount, G->vCount, globalCount, perVertexCount, hashTable, 0, G->uCount + G->vCount);
//         HRR(hipDeviceSynchronize());
//         exectionTime = wtime() - startTime;
//         cout << *globalCount << ' ' << exectionTime << endl;
//         *globalCount = 0;
//     }
//     if (0) //hash>32
//     {
//         startTime = wtime();
//         hashBasedButterflyCounting<<<numBlocks, 1024>>>(D_beginPos, D_edgeList, G->uCount, G->vCount, globalCount, perVertexCount, hashTable, 0, G->breakVertex32);
//         HRR(hipDeviceSynchronize());
//         exectionTime = wtime() - startTime;
//         cout << ' ' << exectionTime;
//     }
//     if (0) //hash<32
//     {
//         startTime = wtime();
//         hashBasedButterflyCounting<<<numBlocks, 1024>>>(D_beginPos, D_edgeList, G->uCount, G->vCount, globalCount, perVertexCount, hashTable, G->breakVertex32, G->uCount + G->vCount);
//         HRR(hipDeviceSynchronize());
//         exectionTime = wtime() - startTime;
//         cout << ' ' << exectionTime;
//     }
//     if (0) //sort<32
//     {
//         startTime = wtime();
//         sortBasedButterflyCounting<<<numBlocks, 1024>>>(D_beginPos, D_edgeList, G->uCount, G->vCount, globalCount, perVertexCount, hashTable, G->breakVertex32, G->uCount + G->vCount);
//         HRR(hipDeviceSynchronize());
//         exectionTime = wtime() - startTime;
//         cout << ' ' << exectionTime;
//     }
//     if (0) //10<hash<32
//     {
//         startTime = wtime();
//         hashBasedButterflyCounting<<<numBlocks, 1024>>>(D_beginPos, D_edgeList, G->uCount, G->vCount, globalCount, perVertexCount, hashTable, G->breakVertex32, G->breakVertex10);
//         HRR(hipDeviceSynchronize());
//         exectionTime = wtime() - startTime;
//         cout << ' ' << exectionTime;
//     }
// }

// if (0) //combined several method
// {
//     startTime = wtime();
//     hashBasedButterflyCounting<<<numBlocks, 1024>>>(D_beginPos, D_edgeList, G->uCount, G->vCount, globalCount, perVertexCount, hashTable, 0, G->breakVertex10);
//     HRR(hipDeviceSynchronize());
//     exectionTime = wtime() - startTime;
//     cout << *globalCount << ' ' << exectionTime << endl;

//     startTime = wtime();
//     heapBasedButterflyCounting<<<numBlocks, 128>>>(D_beginPos, D_edgeList, G->uCount, G->vCount, globalCount, perVertexCount, hashTable, G->breakVertex10, G->uCount + G->vCount);
//     HRR(hipDeviceSynchronize());
//     exectionTime = wtime() - startTime;
//     cout << *globalCount << ' ' << exectionTime << endl;
//     *globalCount = 0;
// }

// *globalCount=0;
// startTime=wtime();
// sortBasedButterflyCounting<<<numBlocks,1024>>>(D_beginPos,D_edgeList,G->uCount,G->vCount,globalCount,perVertexCount2,hashTable,G->breakVertex32,G->vertexCount);
// HRR(hipDeviceSynchronize());
// exectionTime=wtime()-startTime;
// cout<<*globalCount<<' '<<exectionTime;
// for (int i=G->breakVertex32;i<G->breakVertex10;i++)
//     if (perVertexCount[i]!=perVertexCount2[i])
//         cout<<i<<endl;