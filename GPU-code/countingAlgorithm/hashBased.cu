#include "hip/hip_runtime.h"
#include <iostream>
#include <hipcub/hipcub.hpp>
#include <cub/util_type.cuh>
using namespace std;

__device__ 
void hashBasedPerVertexCounting(int vertex, long long *beginPos, int *edgeList, int* hashTable, int uCount, int vCount, unsigned long long *count)
{
    int vertexDegree=beginPos[vertex+1]-beginPos[vertex];

    // put the two hop neighbor of vertex into hash map
    for (int oneHopNeighborID=beginPos[vertex+1]+threadIdx.x/32-32; oneHopNeighborID>=beginPos[vertex]; oneHopNeighborID-=32)
    {
        int oneHopNeighbor=edgeList[oneHopNeighborID];
        if (oneHopNeighbor<=vertex) break;
        for (int twoHopNeighborID=beginPos[oneHopNeighbor+1]+threadIdx.x%32-32; twoHopNeighborID>=beginPos[oneHopNeighbor]; twoHopNeighborID-=32)
        {
            int twoHopNeighbor=edgeList[twoHopNeighborID];
            if (twoHopNeighbor<=vertex) break;
            // if (vertex==1241027)
            //     printf("%d ",twoHopNeighbor);
            *count+=atomicAdd(&hashTable[twoHopNeighbor+blockIdx.x*(uCount+vCount)],1);
            // hashTable[twoHopNeighbor+blockIdx.x*(uCount+vCount)]++;
        }
    }
    __syncthreads();

    // reset the hash map
    if (vertexDegree*vertexDegree>uCount+vCount) //choose the lower costs method 
    {    
        int start=0,end=uCount+vCount;
        start+=blockIdx.x*(uCount+vCount),end+=blockIdx.x*(uCount+vCount);
        for (int i=start+threadIdx.x; i<end; i+=blockDim.x)
        {
            hashTable[i]=0;
        }
    } 
    else
    {
        for (int oneHopNeighborID=beginPos[vertex+1]+threadIdx.x/32-32; oneHopNeighborID>=beginPos[vertex]; oneHopNeighborID-=32)
        {
            int oneHopNeighbor=edgeList[oneHopNeighborID];
            if (oneHopNeighbor<=vertex) break;        
            for (int twoHopNeighborID=beginPos[oneHopNeighbor+1]+threadIdx.x%32-32; twoHopNeighborID>=beginPos[oneHopNeighbor]; twoHopNeighborID-=32)
            {
                int twoHopNeighbor=edgeList[twoHopNeighborID];
                if (twoHopNeighbor<=vertex) break;
                hashTable[twoHopNeighbor+blockIdx.x*(uCount+vCount)]=0;
            }
        }
    }
    __syncthreads();
}

__global__ 
void hashBasedButterflyCounting(long long *beginPos, int *edgeList, int uCount, int vCount, unsigned long long* globalCount, int* perVertexCount, int* hashTable, int startVertex, int endVertex)
{
    __shared__ unsigned long long sharedCount;
    if (threadIdx.x==0) sharedCount=0;
    unsigned long long count=0;
    for (int i=threadIdx.x+blockIdx.x*(uCount+vCount); i<(1+blockIdx.x)*(uCount+vCount); i+=blockDim.x)
    {
        hashTable[i]=0;
    }
    __syncthreads();
    for (int vertex=startVertex+blockIdx.x; vertex<endVertex; vertex+=gridDim.x)
    {
        // count=0;
        // perVertexCount[vertex]=0;
        hashBasedPerVertexCounting(vertex, beginPos, edgeList, hashTable, uCount, vCount, &count);
        // atomicAdd(&perVertexCount[vertex],count);
        // __syncthreads();
    }

    atomicAdd(&sharedCount,count);
    __syncthreads();
    if (threadIdx.x==0) atomicAdd(globalCount,sharedCount);
}