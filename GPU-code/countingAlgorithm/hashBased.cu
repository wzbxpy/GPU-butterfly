#include "hip/hip_runtime.h"
#include <iostream>
#include <hipcub/hipcub.hpp>
#include <cub/util_type.cuh>
using namespace std;
#define FULL_MASK 0xffffffff

__device__ void hashBasedPerVertexCounting(int vertex, long long *beginPos, int *edgeList, int *hashTable, int uCount, int vCount, unsigned long long *count)
{
    int vertexDegree = beginPos[vertex + 1] - beginPos[vertex];

    // put the two hop neighbor of vertex into hash map
    for (int oneHopNeighborID = beginPos[vertex + 1] + threadIdx.x / 32 - 32; oneHopNeighborID >= beginPos[vertex]; oneHopNeighborID -= 32)
    {
        int oneHopNeighbor = edgeList[oneHopNeighborID];
        if (oneHopNeighbor <= vertex)
            break;
        for (int twoHopNeighborID = beginPos[oneHopNeighbor + 1] + threadIdx.x % 32 - 32; twoHopNeighborID >= beginPos[oneHopNeighbor]; twoHopNeighborID -= 32)
        {
            int twoHopNeighbor = edgeList[twoHopNeighborID];
            if (twoHopNeighbor <= vertex)
                break;
            // if (vertex==1241027)
            //     printf("%d ",twoHopNeighbor);
            *count += atomicAdd(&hashTable[twoHopNeighbor + blockIdx.x * (uCount + vCount)], 1);
            // hashTable[twoHopNeighbor+blockIdx.x*(uCount+vCount)]++;
        }
    }
    __syncthreads();

    // reset the hash map
    if (vertexDegree * vertexDegree > uCount + vCount) // choose the lower costs method
    {
        int start = 0, end = uCount + vCount;
        start += blockIdx.x * (uCount + vCount), end += blockIdx.x * (uCount + vCount);
        for (int i = start + threadIdx.x; i < end; i += blockDim.x)
        {
            hashTable[i] = 0;
        }
    }
    else
    {
        for (int oneHopNeighborID = beginPos[vertex + 1] + threadIdx.x / 32 - 32; oneHopNeighborID >= beginPos[vertex]; oneHopNeighborID -= 32)
        {
            int oneHopNeighbor = edgeList[oneHopNeighborID];
            if (oneHopNeighbor <= vertex)
                break;
            for (int twoHopNeighborID = beginPos[oneHopNeighbor + 1] + threadIdx.x % 32 - 32; twoHopNeighborID >= beginPos[oneHopNeighbor]; twoHopNeighborID -= 32)
            {
                int twoHopNeighbor = edgeList[twoHopNeighborID];
                if (twoHopNeighbor <= vertex)
                    break;
                hashTable[twoHopNeighbor + blockIdx.x * (uCount + vCount)] = 0;
            }
        }
    }
    __syncthreads();
}

__device__ void hashBasedPerVertexCounting_newDirection(int vertex, long long *beginPos, int *edgeList, int *hashTable, int uCount, int vCount, unsigned long long *count)
{
    __shared__ int nextOneHopNeighborID;
    int threadId = threadIdx.x % 32;
    int warpId = threadIdx.x / 32;
    int vertexDegree = beginPos[vertex + 1] - beginPos[vertex];

    if (threadIdx.x == 0)
        nextOneHopNeighborID = beginPos[vertex] + 32;
    __syncthreads();
    // put the two hop neighbor of vertex into hash map
    for (int oneHopNeighborID = beginPos[vertex] + threadIdx.x / 32; oneHopNeighborID < beginPos[vertex + 1];)
    {
        int oneHopNeighbor = edgeList[oneHopNeighborID];
        int bound = vertex < oneHopNeighbor ? vertex : oneHopNeighbor;
        for (int twoHopNeighborID = beginPos[oneHopNeighbor] + threadIdx.x % 32; twoHopNeighborID < beginPos[oneHopNeighbor + 1]; twoHopNeighborID += 32)
        {
            int twoHopNeighbor = edgeList[twoHopNeighborID];
            if (twoHopNeighbor >= bound)
                break;
            *count += atomicAdd(&hashTable[twoHopNeighbor + blockIdx.x * (long long)(uCount + vCount)], 1);
        }
        if (threadId == 0)
            oneHopNeighborID = atomicAdd(&nextOneHopNeighborID, 1);
        __syncwarp();

        oneHopNeighborID = __shfl_sync(FULL_MASK, oneHopNeighborID, 0);
    }
    __syncthreads();

    // reset the hash map
    if (vertexDegree * vertexDegree / 1000 > uCount + vCount) // choose the lower costs method
    {
        long long start = 0, end = uCount + vCount;
        start += blockIdx.x * (long long)(uCount + vCount), end += blockIdx.x * (long long)(uCount + vCount);
        for (long long i = start + threadIdx.x; i < end; i += blockDim.x)
        {
            hashTable[i] = 0;
        }
    }
    else
    {
        if (threadIdx.x == 0)
            nextOneHopNeighborID = beginPos[vertex] + 32;
        __syncthreads();
        for (int oneHopNeighborID = beginPos[vertex] + threadIdx.x / 32; oneHopNeighborID < beginPos[vertex + 1];)
        {
            int oneHopNeighbor = edgeList[oneHopNeighborID];
            int bound = vertex < oneHopNeighbor ? vertex : oneHopNeighbor;
            for (int twoHopNeighborID = beginPos[oneHopNeighbor] + threadIdx.x % 32; twoHopNeighborID < beginPos[oneHopNeighbor + 1]; twoHopNeighborID += 32)
            {
                int twoHopNeighbor = edgeList[twoHopNeighborID];
                if (twoHopNeighbor >= bound)
                    break;
                hashTable[twoHopNeighbor + blockIdx.x * (long long)(uCount + vCount)] = 0;
            }
            if (threadId == 0)
                oneHopNeighborID = atomicAdd(&nextOneHopNeighborID, 1);
            __syncwarp();

            oneHopNeighborID = __shfl_sync(FULL_MASK, oneHopNeighborID, 0);
        }
    }
    __syncthreads();
}

__global__ void hashBasedButterflyCounting(long long *beginPos, int *edgeList, int uCount, int vCount, unsigned long long *globalCount, int *perVertexCount, int *hashTable, int startVertex, int endVertex, int *nextVertex)
{
    __shared__ int nextVertexshared;
    __shared__ unsigned long long sharedCount;
    if (threadIdx.x == 0)
        sharedCount = 0;
    unsigned long long count = 0;
    for (long long i = threadIdx.x + blockIdx.x * (uCount + vCount); i < (1 + blockIdx.x) * (uCount + vCount); i += blockDim.x)
    {
        hashTable[i] = 0;
    }
    __syncthreads();
    for (int vertex = startVertex + blockIdx.x; vertex < endVertex;)
    {
        // count=0;
        // perVertexCount[vertex]=0;
        hashBasedPerVertexCounting_newDirection(vertex, beginPos, edgeList, hashTable, uCount, vCount, &count);
        if (threadIdx.x == 0)
            nextVertexshared = atomicAdd(nextVertex, 1);
        __syncthreads();
        vertex = nextVertexshared;
        // vertex += gridDim.x;
        // if (threadIdx.x + blockIdx.x == 0)
        //     printf("%d\n", vertex);
        // atomicAdd(&perVertexCount[vertex],count);
        // __syncthreads();
    }

    atomicAdd(&sharedCount, count);
    __syncthreads();
    if (threadIdx.x == 0)
        atomicAdd(globalCount, sharedCount);
}