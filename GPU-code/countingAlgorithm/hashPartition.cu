#include "hip/hip_runtime.h"
#include <iostream>
#include <hipcub/hipcub.hpp>
#include <cub/util_type.cuh>
using namespace std;

__device__ void hashBasedPerVertexWithPartition(int vertex, long long *beginPosFirst, int *edgeListFirst, long long *beginPosSecond, long long *endPosSecond, int *edgeListSecond, int *hashTable, int length, unsigned long long *count, int partitionNum, int decNum)
{
    int vertexDegree = beginPosFirst[vertex + 1] - beginPosFirst[vertex];

    // put the two hop neighbor of vertex into hash map
    for (int oneHopNeighborID = beginPosFirst[vertex] + threadIdx.x / 32; oneHopNeighborID < beginPosFirst[vertex + 1]; oneHopNeighborID += 32)
    {
        int oneHopNeighbor = edgeListFirst[oneHopNeighborID];
        int bound = vertex < oneHopNeighbor ? vertex : oneHopNeighbor;
        for (int twoHopNeighborID = beginPosSecond[oneHopNeighbor] + threadIdx.x % 32; twoHopNeighborID < beginPosSecond[oneHopNeighbor + 1]; twoHopNeighborID += 32)
        {
            int twoHopNeighbor = edgeListSecond[twoHopNeighborID];
            if (twoHopNeighbor >= bound)
                break;
            *count += atomicAdd(&hashTable[(twoHopNeighbor / partitionNum) - decNum + blockIdx.x * (length)], 1);
        }
    }
    __syncthreads();

    // reset the hash map
    if (vertexDegree * vertexDegree > length) //choose the lower costs method
    // if (1)
    {
        int start = 0, end = length;
        start += blockIdx.x * (length), end += blockIdx.x * (length);
        for (int i = start + threadIdx.x; i < end; i += blockDim.x)
        {
            hashTable[i] = 0;
        }
    }
    else
    {
        for (int oneHopNeighborID = beginPosFirst[vertex] + threadIdx.x / 32; oneHopNeighborID < beginPosFirst[vertex + 1]; oneHopNeighborID += 32)
        {
            int oneHopNeighbor = edgeListFirst[oneHopNeighborID];
            int bound = vertex < oneHopNeighbor ? vertex : oneHopNeighbor;
            for (int twoHopNeighborID = beginPosSecond[oneHopNeighbor] + threadIdx.x % 32; twoHopNeighborID < beginPosSecond[oneHopNeighbor + 1]; twoHopNeighborID += 32)
            {
                int twoHopNeighbor = edgeListSecond[twoHopNeighborID];
                if (twoHopNeighbor >= bound)
                    break;
                hashTable[(twoHopNeighbor / partitionNum) - decNum + blockIdx.x * (length)] = 0;
            }
        }
    }
    __syncthreads();
}

__global__ void hashPartition(long long *beginPosFirst, int *edgeListFirst, long long *beginPosSecond, int *edgeListSecond, unsigned long long *globalCount, int *perVertexCount, int *hashTable, int startVertex, int endVertex, int length, int partitionNum)
{
    __shared__ unsigned long long sharedCount;
    if (threadIdx.x == 0)
        sharedCount = 0;
    unsigned long long count = 0;
    for (int i = threadIdx.x + blockIdx.x * (length); i < (1 + blockIdx.x) * (length); i += blockDim.x)
    {
        hashTable[i] = 0;
    }
    __syncthreads();
    for (int vertex = startVertex + blockIdx.x; vertex < endVertex; vertex += gridDim.x)
    {
        hashBasedPerVertexWithPartition(vertex, beginPosFirst, edgeListFirst, beginPosSecond, beginPosSecond + 1, edgeListSecond, hashTable, length, &count, partitionNum, 0);
    }

    atomicAdd(&sharedCount, count);
    __syncthreads();
    if (threadIdx.x == 0)
        atomicAdd(globalCount, sharedCount);
}