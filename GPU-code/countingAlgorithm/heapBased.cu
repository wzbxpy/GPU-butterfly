#include "hip/hip_runtime.h"
#include <iostream>
#include <hipcub/hipcub.hpp>
#include <cub/util_type.cuh>
using namespace std;

struct heap
{
    int element;
    int* now;
    int* end;
};

__device__ int* binarySearch(int* a, int* b, int x)
{
    while (a<b)
    {
        int* mid=a+((b-a)/2);
        if (*mid<=x) a=mid+1; else b=mid;
    }
    return a;
}

__device__ 
void heapBasedPerVertexCounting(int vertex, long long *beginPos, int *edgeList, int* hashTable, int uCount, int vCount, unsigned long long *count)
{
    struct heap H[11];
    H[0].element=-1;
    int k=1;
    
    // first creat the heap 
    for (int oneHopNeighborID=beginPos[vertex+1]-1; oneHopNeighborID>=beginPos[vertex]; oneHopNeighborID--)
    {
        int oneHopNeighbor=edgeList[oneHopNeighborID];
        if (oneHopNeighbor<=vertex) break;
        // get the first neighbor in each oneHopNeighbor's neighbor list 
        H[k].end=edgeList+beginPos[oneHopNeighbor+1];
        H[k].now=binarySearch(edgeList+beginPos[oneHopNeighbor], H[k].end, vertex);
        if (H[k].now>=H[k].end)  continue;
        H[k].element=*H[k].now;

        // update the heap
        int p=k;
        while (H[p].element<H[p/2].element)
        {
            struct heap t;
            t=H[p];
            H[p]=H[p/2];
            p/=2;
            H[p]=t;
        }
        k++;
    }
    int previousElement=-1,cc=1;
    // second pop the top element in heap and add new element from its corresponding neighbor list
    for(k--;k>=1;)
    {
        // update the count of butterflies
        int nowElement=H[1].element;
        if (nowElement==previousElement) {cc++;}
        else { *count+=cc*(cc-1)/2; cc=1; previousElement=nowElement;}

        // add the next element into heap
        H[1].now++;
        if (H[1].now>=H[1].end) 
        {
            struct heap t;
            t=H[1];
            H[1]=H[k];
            H[k]=t;
            k--;
        }
        else H[1].element=*(H[1].now);

        // update the heap
        int p=1;
        for(;;)
        {
            if (p*2>k) break;
            p*=2;
            if (p+1<=k&&H[p].element>H[p+1].element) p++;
            if (H[p].element<H[p/2].element)
            {
                struct heap t;
                t=H[p];
                H[p]=H[p/2];
                H[p/2]=t;
            }
            else
            break;
        }
        

    }
    *count+=cc*(cc-1)/2; // the last series of element need to be added
}


__global__ 
void heapBasedButterflyCounting(long long *beginPos, int *edgeList, int uCount, int vCount, unsigned long long* globalCount, int* perVertexCount, int* hashTable, int startVertex, int endVertex)
{
    __shared__ unsigned long long sharedCount;
    if (threadIdx.x==0) sharedCount=0;
    unsigned long long count=0;
    for (int vertex=startVertex+blockIdx.x*blockDim.x+threadIdx.x; vertex<endVertex; vertex+=gridDim.x*blockDim.x)
    {
        // count=0;
        heapBasedPerVertexCounting(vertex, beginPos, edgeList, hashTable, uCount, vCount, &count);
        // perVertexCount[vertex]=count;
    }
    atomicAdd(&sharedCount,count);
    __syncthreads();
    if (threadIdx.x==0) atomicAdd(globalCount,sharedCount);
}
