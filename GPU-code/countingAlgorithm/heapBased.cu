#include "hip/hip_runtime.h"
#include <iostream>
#include <hipcub/hipcub.hpp>
#include <cub/util_type.cuh>
#ifdef DEBUG
#define DBGprint(...) printf(__VA_ARGS__)
#else
#define DBGprint(...)
#endif
#define MAXINT 2147483641
#define FULL_MASK 0xffffffff
using namespace std;

struct heap
{
    int element;
    int *now;
    int *end;
};

__device__ int *binarySearch(int *a, int *b, int x)
{
    while (a < b)
    {
        int *mid = a + ((b - a) / 2);
        if (*mid < x)
            a = mid + 1;
        else
            b = mid;
    }
    return a;
}

__device__ void heapBasedPerVertexCounting(int vertex, long long *beginPos, int *edgeList, int *hashTable, int uCount, int vCount, unsigned long long *count)
{
    struct heap H[11];
    H[0].element = -1;
    int k = 1;

    // first creat the heap
    for (int oneHopNeighborID = beginPos[vertex]; oneHopNeighborID < beginPos[vertex + 1]; oneHopNeighborID++)
    {
        int oneHopNeighbor = edgeList[oneHopNeighborID];
        // get the first neighbor in each oneHopNeighbor's neighbor list
        H[k].now = edgeList + beginPos[oneHopNeighbor];
        H[k].end = binarySearch(H[k].now, edgeList + beginPos[oneHopNeighbor + 1], vertex < oneHopNeighbor ? vertex : oneHopNeighbor);
        // printf("%d %d %d\n", *H[k].now, *H[k].end, vertex < oneHopNeighbor ? vertex : oneHopNeighbor);
        // for (int *aaaaa = H[k].now; aaaaa < H[k].end; aaaaa++)
        // {
        //     DBGprint("%d ", *aaaaa);
        // }
        // DBGprint("\n");
        if (H[k].now >= H[k].end)
            continue;
        H[k].element = *H[k].now;
        // update the heap
        int p = k;
        while (H[p].element < H[p / 2].element)
        {
            struct heap t;
            t = H[p];
            H[p] = H[p / 2];
            p /= 2;
            H[p] = t;
        }
        k++;
    }
    int previousElement = -1, cc = 1;
    // second pop the top element in heap and add new element from its corresponding neighbor list
    for (--k; k >= 1;)
    {
        // update the count of butterflies
        int nowElement = H[1].element;

        // if (iddd > 967 && iddd < 974)
        // {
        //     DBGprint("%d,,%d,,%d,,%d\n", cc, nowElement, previousElement, iddd);
        //     for (int aaaaa = 0; aaaaa < k; aaaaa++)
        //         DBGprint("%d ", H[aaaaa].element);
        //     DBGprint("\n");
        // }
        if (nowElement == previousElement)
        {
            cc++;
        }
        else
        {
            // if (cc > 1)
            //     DBGprint("%d,%d\n", previousElement, cc);
            *count += cc * (cc - 1) / 2;
            cc = 1;
            previousElement = nowElement;
        }

        // add the next element into heap
        H[1].now++;
        if (H[1].now >= H[1].end)
        {
            struct heap t;
            t = H[1];
            H[1] = H[k];
            H[k] = t;
            k--;
        }
        else
            H[1].element = *(H[1].now);

        // update the heap
        int p = 1;
        for (;;)
        {
            if (p * 2 > k)
                break;
            p *= 2;
            if (p + 1 <= k && H[p].element > H[p + 1].element)
                p++;
            if (H[p].element < H[p / 2].element)
            {
                struct heap t;
                t = H[p];
                H[p] = H[p / 2];
                H[p / 2] = t;
            }
            else
                break;
        }
    }
    DBGprint("%d\n", cc);
    *count += cc * (cc - 1) / 2; // the last series of element need to be added
}

__global__ void heapBasedButterflyCounting(long long *beginPos, int *edgeList, int uCount, int vCount, unsigned long long *globalCount, int *perVertexCount, int *hashTable, int startVertex, int endVertex)
{
    __shared__ unsigned long long sharedCount;
    if (threadIdx.x == 0)
        sharedCount = 0;
    unsigned long long count = 0;
    for (int vertex = startVertex + blockIdx.x * blockDim.x + threadIdx.x; vertex < endVertex; vertex += gridDim.x * blockDim.x)
    {
        // count=0;
        heapBasedPerVertexCounting(vertex, beginPos, edgeList, hashTable, uCount, vCount, &count);
        // perVertexCount[vertex]=count;
    }
    atomicAdd(&sharedCount, count);
    __syncthreads();
    if (threadIdx.x == 0)
        atomicAdd(globalCount, sharedCount);
}

__device__ void heapBasedPerVertexCounting_byWarp(int vertex, long long *beginPos, int *edgeList, int *hashTable, int uCount, int vCount, unsigned long long *count)
{

    int warpID = threadIdx.x / 32;
    int threadId = threadIdx.x % 32;
    struct heap h;
    h.element = MAXINT;
    int k = 1;

    // first creat the heap
    for (int oneHopNeighborID = beginPos[vertex] + threadId; oneHopNeighborID < beginPos[vertex + 1]; oneHopNeighborID += 32)
    {
        int oneHopNeighbor = edgeList[oneHopNeighborID];
        // get the first neighbor in each oneHopNeighbor's neighbor list
        h.now = edgeList + beginPos[oneHopNeighbor];
        h.end = binarySearch(h.now, edgeList + beginPos[oneHopNeighbor + 1], vertex < oneHopNeighbor ? vertex : oneHopNeighbor);
        if (h.now < h.end)
            h.element = *(h.now);
    }
    int previousElement = -1, cc = 1;
    // second pop the top element in heap and add new element from its corresponding neighbor list
    for (;;)
    {
        int element = h.element;

        int id = threadId;
        int needBreak = 0;
        for (int offset = 16; offset > 0; offset /= 2)
        {
            int otherElement = __shfl_down_sync(FULL_MASK, element, offset);
            int otherId = __shfl_down_sync(FULL_MASK, id, offset);
            if (otherElement < element)
            {
                element = otherElement;
                id = otherId;
            }
        }
        __syncwarp();

        if (threadId == 0)
        {
            // printf("%d %d\n", element, id);
            if (element == MAXINT)
            {
                needBreak = 1;
            }
            else if (element == previousElement)
            {
                cc++;
            }
            else
            {
                *count += cc * (cc - 1) / 2;
                cc = 1;
                previousElement = element;
            }
        }
        __syncwarp();
        needBreak = __shfl_sync(FULL_MASK, needBreak, 0);
        if (needBreak)
        {
            break;
        }
        id = __shfl_sync(FULL_MASK, id, 0);

        if (threadId == id)
        {
            h.now++;
            if (h.now < h.end)
                h.element = *(h.now);
            else
                h.element = MAXINT;
        }
    }
    *count += cc * (cc - 1) / 2; // the last series of element need to be added
}

__global__ void heapBasedButterflyCounting_byWarp(long long *beginPos, int *edgeList, int uCount, int vCount, unsigned long long *globalCount, int *perVertexCount, int *hashTable, int startVertex, int endVertex)
{
    __shared__ unsigned long long sharedCount;
    if (threadIdx.x == 0)
        sharedCount = 0;
    unsigned long long count = 0;
    int warpID = threadIdx.x / 32;
    int warpNum = blockDim.x / 32;
    for (int vertex = startVertex + blockIdx.x * warpNum + warpID; vertex < endVertex; vertex += gridDim.x * warpNum)
    {
        // count=0;
        heapBasedPerVertexCounting_byWarp(vertex, beginPos, edgeList, hashTable, uCount, vCount, &count);
        // perVertexCount[vertex]=count;
    }
    if (threadIdx.x % 32 == 0)
        atomicAdd(&sharedCount, count);
    __syncthreads();
    if (threadIdx.x == 0)
        atomicAdd(globalCount, sharedCount);
}