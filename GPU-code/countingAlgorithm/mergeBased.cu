#include "hip/hip_runtime.h"
#include <iostream>
#include "../globalPara.h"
#ifdef DEBUG
#define DBGprint(...) printf(__VA_ARGS__)
#else
#define DBGprint(...)
#endif
#define MAXINT 2147483641
#define FULL_MASK 0xffffffff
using namespace std;
#define subwarpSize 8
#define subwarpNum (32 / subwarpSize)
#define sharedElementSize blockSize *subwarpSize

struct marker
{
    int element;
    int *globalNow;
    int localNow;
    int len;
};
__device__ void mergeBasedPerVertexCounting__backup(int vertex, long long *beginPos, int *edgeList, int *hashTable, int uCount, int vCount, unsigned long long *count)
{

    int warpId = threadIdx.x / 32;
    int threadId = threadIdx.x % 32;
    struct marker h;
    h.element = MAXINT;
    int bound;

    // first creat the marker
    for (int oneHopNeighborID = beginPos[vertex] + threadId; oneHopNeighborID < beginPos[vertex + 1]; oneHopNeighborID += 32)
    {
        int oneHopNeighbor = edgeList[oneHopNeighborID];
        // get the first neighbor in each oneHopNeighbor's neighbor list
        bound = vertex < oneHopNeighbor ? vertex : oneHopNeighbor;

        int start = beginPos[oneHopNeighbor];
        int end = beginPos[oneHopNeighbor + 1];
        h.globalNow = edgeList + start;
        h.len = end - start;
        if (h.len > 0)
        {
            int element = *(h.globalNow);
            if (element < bound)
                h.element = element;
            else
                h.len = -1;
            h.len--;
            h.globalNow++;
        }
    }
    int previousElement = -1, cc = 1;
    // second pop the top element in marker and add new element from its corresponding neighbor list
    for (;;)
    {
        int element = __reduce_min_sync(FULL_MASK, h.element);
        if (element == MAXINT)
            break;
        int matched = element == h.element;
        if (matched)
        {
            h.element = MAXINT;
            if (h.len > 0)
            {
                int element = *(h.globalNow);
                if (element < bound)
                    h.element = element;
                else
                    h.len = -1;
                h.len--;
                h.globalNow++;
            }
        }
        matched = __reduce_add_sync(FULL_MASK, matched);
        if (threadId == 0)
            *count += matched * (matched - 1) / 2;
    }
}

__device__ void mergeBasedPerVertexCounting__activethread(int vertex, long long *beginPos, int *edgeList, int *hashTable, int uCount, int vCount, unsigned long long *count)
{

    int warpId = threadIdx.x / 32;
    int threadId = threadIdx.x % 32;
    struct marker h;
    h.element = MAXINT;
    int bound;
    __shared__ int activeThreads[blockSize / 32];

    // first creat the marker
    for (int oneHopNeighborID = beginPos[vertex] + threadId; oneHopNeighborID < beginPos[vertex + 1]; oneHopNeighborID += 32)
    {
        int oneHopNeighbor = edgeList[oneHopNeighborID];
        // get the first neighbor in each oneHopNeighbor's neighbor list
        bound = vertex < oneHopNeighbor ? vertex : oneHopNeighbor;

        int start = beginPos[oneHopNeighbor];
        int end = beginPos[oneHopNeighbor + 1];
        h.globalNow = edgeList + start;
        h.len = end - start;
        if (h.len > 0)
        {
            int element = *(h.globalNow);
            if (element < bound)
                h.element = element;
            else
                h.len = -1;
            h.len--;
            h.globalNow++;
        }
    }

    __syncwarp();
    int isActive = h.element < MAXINT;
    isActive = __reduce_add_sync(FULL_MASK, isActive);
    if (threadId == 0)
        activeThreads[warpId] = isActive;
    int previousElement = -1, cc = 1;
    __syncwarp();
    // second pop the top element in marker and add new element from its corresponding neighbor list
    for (; activeThreads[warpId] > 1;)
    {
        int element = __reduce_min_sync(FULL_MASK, h.element);
        if (element == MAXINT)
            break;
        int matched = element == h.element;
        if (matched)
        {
            h.element = MAXINT;
            if (h.len > 0)
            {
                int element = *(h.globalNow);
                if (element < bound)
                    h.element = element;
                else
                    h.len = -1;
                h.len--;
                if (h.len < 0)
                    atomicAdd(activeThreads + warpId, -1);
                h.globalNow++;
            }
        }
        matched = __reduce_add_sync(FULL_MASK, matched);
        if (threadId == 0)
            *count += matched * (matched - 1) / 2;
    }
}

__global__ void mergeBasedButterflyCounting(long long *beginPos, int *edgeList, int uCount, int vCount, unsigned long long *globalCount, int *perVertexCount, int *hashTable, int startVertex, int endVertex)
{
    __shared__ unsigned long long sharedCount;
    if (threadIdx.x == 0)
        sharedCount = 0;
    unsigned long long count = 0;
    int warpId = threadIdx.x / 32;
    int warpNum = blockDim.x / 32;
    for (int vertex = startVertex + blockIdx.x * warpNum + warpId; vertex < endVertex; vertex += gridDim.x * warpNum)
    {
        // count=0;
        mergeBasedPerVertexCounting__backup(vertex, beginPos, edgeList, hashTable, uCount, vCount, &count);
        // perVertexCount[vertex]=count;
    }
    if (threadIdx.x % 32 == 0)
    {
        atomicAdd(&sharedCount, count);
    }
    __syncthreads();
    if (threadIdx.x == 0)
    {
        atomicAdd(globalCount, sharedCount);
    }
}
