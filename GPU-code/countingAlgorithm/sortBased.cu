#include "hip/hip_runtime.h"
#include <iostream>
#include <hipcub/hipcub.hpp>
#include <cub/util_type.cuh>
#define infinit 99999999

using namespace std;

__device__ 
void sortBasedPerVertexCounting(int vertex, long long *beginPos, int *edgeList, int* hashTable, int uCount, int vCount, unsigned long long *count)
{
    
    // int vertexDegree=beginPos[vertex+1]-beginPos[vertex];
    int ele[1],k=0;
    for (int i=0;i<1;i++)
        ele[i]=infinit;
    for (int oneHopNeighborID=beginPos[vertex+1]+threadIdx.x/32-32; oneHopNeighborID>=beginPos[vertex]; oneHopNeighborID-=32)
    {
        int oneHopNeighbor=edgeList[oneHopNeighborID];
        if (oneHopNeighbor<=vertex) break;
        for (int twoHopNeighborID=beginPos[oneHopNeighbor+1]+threadIdx.x%32-32; twoHopNeighborID>=beginPos[oneHopNeighbor]; twoHopNeighborID-=32)
        {
            int twoHopNeighbor=edgeList[twoHopNeighborID];
            if (twoHopNeighbor<=vertex) break;
            ele[k]=twoHopNeighbor;
            // if (threadIdx.x==1023&&vertex==1241027) 
            // {
            //     printf("vertexdegree=%d,onehopdegree=%d\ntwoHopNeighborID=%d,oneHopNeighborID=%d\n",beginPos[vertex+1]-beginPos[vertex],beginPos[oneHopNeighbor+1]-beginPos[oneHopNeighbor],twoHopNeighborID-beginPos[oneHopNeighbor],oneHopNeighborID-beginPos[vertex],oneHopNeighbor);
            //     // printf("\n");
            // }
            k++;
        }
    }
    // if(k>1) printf("%d\n",threadIdx.x);
    __syncthreads();
    typedef hipcub::BlockLoad<int, 1024, 1, hipcub::BLOCK_LOAD_TRANSPOSE> BlockLoadT;    
    typedef hipcub::BlockStore<int, 1024, 1, hipcub::BLOCK_STORE_TRANSPOSE> BlockStoreT;
    typedef hipcub::BlockRadixSort<int, 1024, 1> BlockRadixSortT;
    __shared__ union {
        typename BlockLoadT::TempStorage       load; 
        typename BlockStoreT::TempStorage      store; 
        typename BlockRadixSortT::TempStorage  sort;
    } temp_storage; 

    BlockRadixSortT(temp_storage.sort).Sort(ele);
    __syncthreads();    // Barrier for smem reuse
    __shared__ int twoHopNeighborList[1027];
    BlockStoreT(temp_storage.store).Store(twoHopNeighborList+1, ele);    
    __syncthreads();
    twoHopNeighborList[1025]=-1;
    twoHopNeighborList[0]=0;
    // if (threadIdx.x==0&&vertex==1241027) 
    // {
    //     for (int i=0;i<1024;i++)
    //         printf("%d ",twoHopNeighborList[i]);
    //     printf("\n");
    // }
    __syncthreads();
    for (int i=threadIdx.x+1;i<1025;i+=blockDim.x)
        if ((twoHopNeighborList[i]!=infinit)&&(twoHopNeighborList[i]==twoHopNeighborList[i+1])) twoHopNeighborList[i]=i+1;
        else twoHopNeighborList[i]=i;
    __syncthreads();
    for (int k=1;k<=6;k++)
        for (int i=threadIdx.x+1;i<1025;i+=blockDim.x)
            twoHopNeighborList[i]=twoHopNeighborList[twoHopNeighborList[i]];
    __syncthreads();
    for (int i=threadIdx.x+1;i<1025;i+=blockDim.x)
        twoHopNeighborList[i]=twoHopNeighborList[i]-i;
    __syncthreads();

    for (int i=threadIdx.x+1;i<1025;i+=blockDim.x)
        if (twoHopNeighborList[i]>twoHopNeighborList[i-1]) 
            *count+=twoHopNeighborList[i]*(twoHopNeighborList[i]+1)/2;
    __syncthreads();
}

__global__ 
void sortBasedButterflyCounting(long long *beginPos, int *edgeList, int uCount, int vCount, unsigned long long* globalCount, int* perVertexCount, int* hashTable, int startVertex, int endVertex)
{
    __shared__ unsigned long long sharedCount;
    if (threadIdx.x==0) sharedCount=0;
    unsigned long long count=0;

    __syncthreads();
    for (int vertex=startVertex+blockIdx.x; vertex<endVertex; vertex+=gridDim.x)
    {
        // count=0;
        // perVertexCount[vertex]=0;
        // int vertexDegree=beginPos[vertex+1]-beginPos[vertex];
        sortBasedPerVertexCounting(vertex, beginPos, edgeList, hashTable, uCount, vCount, &count);
        // atomicAdd(&perVertexCount[vertex],count);
        // __syncthreads();
    }

    atomicAdd(&sharedCount,count);
    __syncthreads();
    if (threadIdx.x==0) atomicAdd(globalCount,sharedCount);
}