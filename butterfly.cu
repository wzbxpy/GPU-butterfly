#include "hip/hip_runtime.h"
#include <iostream>
#include "graph.h"
#include "wtime.h"
#include "util.h"
#include <hipcub/hipcub.hpp>
#include <cub/util_type.cuh>

#define blocknumber 1

using namespace std;
// using namespace hipcub;

// template <int BLOCK_THREADS, int ITEMS_PER_THREAD>
// __global__ void neighborSorting(int* d_in)
// {
//     int vertex=blockIdx.x;
//     typedef hipcub::BlockLoad<int, BLOCK_THREADS, ITEMS_PER_THREAD, hipcub::BLOCK_LOAD_TRANSPOSE> BlockLoadT;    
//     typedef hipcub::BlockStore<int, BLOCK_THREADS, ITEMS_PER_THREAD, hipcub::BLOCK_STORE_TRANSPOSE> BlockStoreT;
//     typedef hipcub::BlockRadixSort<int, BLOCK_THREADS, ITEMS_PER_THREAD> BlockRadixSortT;
//     __shared__ union {
//         typename BlockLoadT::TempStorage       load; 
//         typename BlockStoreT::TempStorage      store; 
//         typename BlockRadixSortT::TempStorage  sort;
//     } temp_storage; 

//     int thread_keys[ITEMS_PER_THREAD];
//     // int *p;
//     // p=thread_keys;
//     int block_offset = blockIdx.x * (BLOCK_THREADS * ITEMS_PER_THREAD);      
//     BlockLoadT(temp_storage.load).Load(d_in + block_offset, thread_keys);

//     __syncthreads();    // Barrier for smem reuse
//     // Collectively sort the keys
//     BlockRadixSortT(temp_storage.sort).Sort(thread_keys);
//     __syncthreads();    // Barrier for smem reuse
//     // Store the sorted segment 
//     BlockStoreT(temp_storage.store).Store(d_in + block_offset, thread_keys);

// }




__global__ 
void butterflyCounting(long long *beginPos, int *edgeList, int uCount, int vCount, unsigned long long* globalCount, int* hashTable)
{
    __shared__ unsigned long long sharedCount;
    if (threadIdx.x==0) sharedCount=0;
    unsigned long long count=0;

    for (int i=threadIdx.x+blockIdx.x*(uCount+vCount); i<uCount+vCount+blockIdx.x*(uCount+vCount); i+=blockDim.x)
        hashTable[i]=0;
    __syncthreads();
    for (int vertex=blockIdx.x; vertex<uCount+vCount; vertex+=gridDim.x)
    {
        int oneHopNeighborID=threadIdx.x/32+beginPos[vertex];
        int vertexDegree=beginPos[vertex+1]-beginPos[vertex];

        for (; oneHopNeighborID<beginPos[vertex+1]; oneHopNeighborID+=32)
        {
            int oneHopNeighbor=edgeList[oneHopNeighborID];
            int oneHopNeighborDegree=beginPos[oneHopNeighbor+1]-beginPos[oneHopNeighbor];
            if (oneHopNeighborDegree>vertexDegree || (oneHopNeighborDegree==vertexDegree && oneHopNeighbor<=vertex)) continue;
            int twoHopNeighborID=threadIdx.x%32+beginPos[oneHopNeighbor];
            for (; twoHopNeighborID<beginPos[oneHopNeighbor+1]; twoHopNeighborID+=32)
            {
                int twoHopNeighbor=edgeList[twoHopNeighborID];
                int twoHopNeighborDegree=beginPos[twoHopNeighbor+1]-beginPos[twoHopNeighbor];
                if ((twoHopNeighborDegree>vertexDegree) || (twoHopNeighborDegree==vertexDegree && twoHopNeighbor<=vertex)) continue;
                // if (twoHopNeighbor<=vertex) continue;
                // printf("%d %d\n",twoHopNeighborDegree,vertexDegree);
                atomicAdd(&hashTable[twoHopNeighbor+blockIdx.x*(uCount+vCount)],1);
                // count++;
            }
        }
        __syncthreads();
        int start=0,end=uCount;
        if (vertex>=uCount)
        {
            start=uCount,end=uCount+vCount;
        }
        start+=blockIdx.x*(uCount+vCount),end+=blockIdx.x*(uCount+vCount);
        // int start=0,end=uCount+vCount;
        // if (threadIdx.x==0) printf("%d\n",ccc);
        for (int i=start+threadIdx.x; i<end; i+=blockDim.x)
        {
            count+=hashTable[i]*(hashTable[i]-1)/2;
            hashTable[i]=0;
        }
        __syncthreads();
    }



    atomicAdd(&sharedCount,count);
    __syncthreads();
    if (threadIdx.x==0) atomicAdd(globalCount,sharedCount);
}

int BC(graph bipartiteGraph)
{
    
    double startTime,exectionTime;

    long long* D_beginPos;
    int* D_edgeList;
    HRR(hipMalloc((void **) &D_beginPos,sizeof(long long)*(bipartiteGraph.uCount+bipartiteGraph.vCount+1)));
    HRR(hipMalloc((void **) &D_edgeList,sizeof(int)*(bipartiteGraph.edgeCount)));
    HRR(hipMemcpy(D_beginPos,bipartiteGraph.beginPos,sizeof(long long)*(bipartiteGraph.uCount+bipartiteGraph.vCount+1), hipMemcpyHostToDevice));
    HRR(hipMemcpy(D_edgeList,bipartiteGraph.edgeList,sizeof(int)*(bipartiteGraph.edgeCount), hipMemcpyHostToDevice));


    unsigned long long *globalCount;
    unsigned long long count=0;
    HRR(hipMallocManaged(&globalCount, sizeof(unsigned long long)));
    
    int *hashTable;
    HRR(hipMallocManaged(&hashTable, sizeof(int)*(bipartiteGraph.uCount+bipartiteGraph.vCount)*blocknumber));
    HRR(hipMemcpy(globalCount,&count,sizeof(unsigned long long), hipMemcpyHostToDevice));


    startTime=wtime();
    butterflyCounting<<<blocknumber,1024>>>(D_beginPos,D_edgeList,bipartiteGraph.uCount,bipartiteGraph.vCount,globalCount,hashTable);
    HRR(hipDeviceSynchronize());
    exectionTime=wtime()-startTime;
    cout<<*globalCount<<' '<<exectionTime<<endl;

    // int *d_in,*in;
    // int num_blocks=10240;
    // int N=1024*16*num_blocks;
    // cout<<N<<endl;
    // in=new int[N];
    // for (int i=0;i<N;i++)
    // {
    //     in[i]=N-i;
    // }
    // HRR(hipMalloc((void **)&d_in, N*sizeof(int)));
    // HRR(hipMemcpy(d_in,in,sizeof(int)*N, hipMemcpyHostToDevice));

    // neighborSorting<1024, 16><<<num_blocks, 1024>>>(d_in); 

    // HRR(hipDeviceSynchronize());
    // exectionTime=wtime()-startTime;
    
    // HRR(hipMemcpy(in,d_in,sizeof(int)*N, hipMemcpyDeviceToHost));
    // cout<<in[100]<<' '<<exectionTime<<endl;
    




    HRR(hipFree(D_beginPos));
    HRR(hipFree(D_edgeList));
    return 0;
}
